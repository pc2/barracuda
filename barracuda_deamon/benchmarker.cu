/**
 * \file
 * \brief	Benchmarking section
 *
 * @author	Dominic Eschweiler weiler@upb.de
 *
 * Status	: STABLE\n
 * Date of creation : 31.5.2008
 *
 */

/*****************************************************************
 *
 * Barracuda is a experimental microdriver extension to the 
 * linux-kernel that is able to outsource common functions to
 * the userspace. It was intensionally designed to accelerate
 * CPU-intensive Tasks on a GPU.
 *
 * Copyright (C) 2009 Dominic Eschweiler
 *
 * This program is free software; you can redistribute it and/or 
 * modify it under the terms of the GNU General Public License as 
 * published by the Free Software Foundation; either only GPLv2 - 
 * version 2 of the License.
 *
 * This program is distributed in the hope that it will be useful, 
 * but WITHOUT ANY WARRANTY; without even the implied warranty of 
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the 
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public 
 * License along with this program; 
 * if not, see <http://www.gnu.org/licenses/>.
 *
 *****************************************************************/

#include <stdlib.h>
#include <stdio.h>
#include <unistd.h>
#include <string.h>
#include <dirent.h>
#include <sys/stat.h>
#include <sys/file.h>
#include <sys/types.h>
#include <time.h>
#include <sys/time.h>
#include <math.h>

#ifndef NOCUDA
	#include <hip/hip_runtime_api.h>
	# include "bench/cuda_xor_test.h"
	# include "bench/cuda_shift_test.h"
#endif

# include "bench/gen_syndrome_test.h"

# include "benchmarker.h"
# include "service.h"



/**
 * Benchmark main-routine.
 *
 * @param *mode						: what shall we benchmark.
 *									  Valid modes are :\n
 *									  DRYRUN -> for benchmarking the pure implementation speed\n
 * @param gen_syndrome_list[]		: function pointers
 * @param **implemenatation_names	: related names of each function
 * @param number_of_implementations	: # of implementations
 * @param c_mode					: implementation number
 *
 * @returns		void
 */

HOST void baracuda_benchmarker(	char *mode,
								syndrome_func gen_syndrome_list[],
								char **implemenatation_names,
								int number_of_implementations, int c_mode )
{

if( strcmp(mode, "DRYRUN") == 0 ){
	printf("Starting DRYRUN test for testing all implementations.\n");
	compare_all_implementations(	gen_syndrome_list, 
									implemenatation_names,
									c_mode );
	}

#ifndef NOCUDA
if( strcmp(mode, "CUDA_BANDWIDTH") == 0 ){
	printf("Starting CUDA_BANDWIDTH for testing the bandwidth beteween host and cuda device.\n");
	}

if( strcmp(mode, "CUDA_XOR") == 0 ){
	printf("Starting CUDA_XOR for testing the pure XOR performance.\n");
	test_cuda_xor_perf();
	}
	
if( strcmp(mode, "CUDA_SHIFT") == 0 ){
	printf("Starting CUDA_SHIFT for testing the pure SHIFT performance.\n");
	test_cuda_shift_perf();
	}
#endif
	
}
