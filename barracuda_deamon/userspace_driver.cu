#include "hip/hip_runtime.h"
/**
 * \file
 * \brief	This is the userspace driver, which receives function-calls from the 
 *			userspace and delegates it to the choosen RS-Implementation.
 *
 * @author	Dominic Eschweiler weiler@upb.de
 *
 * Status	: STABLE \n
 * Date of creation : 11.9.2008
 *
 */

/*****************************************************************
 *
 * Barracuda is a experimental microdriver extension to the 
 * linux-kernel that is able to outsource common functions to
 * the userspace. It was intensionally designed to accelerate
 * CPU-intensive Tasks on a GPU.
 *
 * Copyright (C) 2009 Dominic Eschweiler
 *
 * This program is free software; you can redistribute it and/or 
 * modify it under the terms of the GNU General Public License as 
 * published by the Free Software Foundation; either only GPLv2 - 
 * version 2 of the License.
 *
 * This program is distributed in the hope that it will be useful, 
 * but WITHOUT ANY WARRANTY; without even the implied warranty of 
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the 
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public 
 * License along with this program; 
 * if not, see <http://www.gnu.org/licenses/>.
 *
 *****************************************************************/

#include <stdlib.h>
#include <stdio.h>
#include <unistd.h>
#include <string.h>
#include <dirent.h>
#include <sys/stat.h>
#include <sys/file.h>
#include <sys/types.h>
#include <time.h>
#include <sys/time.h>
#include <math.h>
#include <fcntl.h>
#include <sched.h>
#include <signal.h>
#include <syslog.h>
#include <sys/mman.h>
#include <linux/netlink.h>
#include <linux/rtnetlink.h>
#include <bits/sockaddr.h>
#include <sys/socket.h>
#include <memory.h>
#include <malloc.h>
#include <sys/ioctl.h>

#ifndef NOCUDA
	#include <hip/hip_runtime_api.h>
	#include "cuda/raid6cuda.h"
#endif

#include "vanilla/raid6vanilla.h"
#include "smp/raid6smp.h"
#include "userspace_driver.h"
#include "definitions.h"
#include "service.h"

void kill_handler(int signum);
void alarm_handler(int signum);

int server_ioctl_callback(syndrome_func gen_syndrome);
int server_netlink(syndrome_func gen_syndrome);
int server_procfs(syndrome_func gen_syndrome);

syndrome_container *copy_act_syndrome_block( void );
void copyback_act_syndrome_block( syndrome_container *smc );

syndrome_container *get_act_syndrome_block( void );
void unget_act_syndrome_block( syndrome_container *smc );

void gen_message_container(struct msghdr *msg);
void destroy_message_container(struct msghdr *msg);
void add_payload(struct msghdr *msg, char *payload );
void get_payload(struct msghdr *msg, char *payload);


/* Variables */
volatile sig_atomic_t keep_going = 1;
volatile static int fd;

static syndrome_container *ret_global;
static int smc_flag = 0;
static void **global_dptrs;

/* Defines */
#define NETLINK_RS_SERVER 25
#define NL_COMMAND 0x11
#define MAX_PAYLOAD 1024
#define IOCTL_GETVALUE 0x0001

/* __Marshalling Method__
 * If this is undefined, the slow mmap method for marshalling is used. If this 
 * is defined, the copy_to_user method is used.
 */
//#define COPY_MARSHALLING


/*MAIN_THREAD_________________________________________________________________*/
/**
 * This is the main function of control which implements the userspace driver.
 * This part of code runs in a detached (demonized) process in the userspace.
 * All outputs at this parts are delegated to the syslog (/var/log /messages),
 * because a simple printf isn't possible on such an process.
 *
 * @param			*rs_function	: Signal number
 *
 * @returns			EXIT_FAILURE on error, EXIT_SUCCESS on no error
 */

int userspace_driver_main(void *rs_function)
{
pid_t pid = getpid();
struct stat status;
thread_container *tc;
int c_mode;
syndrome_func gen_syndrome;

/* Malloc the dptr array */
global_dptrs = (void **)malloc(255 * sizeof(void*));
	
/* reassemble the function pointer and the mode number */	
tc 				= (thread_container *)rs_function;
c_mode			= tc->c_mode;
gen_syndrome	= tc->gen_syndrome;
	
syslog(LOG_NOTICE, "Daemon-Mode called\n");
syslog(LOG_NOTICE, "Connection-Mode is %d\n", c_mode);
	
/* 
 store the pid into a file. This could also be used to look if there is 
 already baracuda process.
 */

if( stat("/tmp/baracuda_pid", &status) == 0 ){
	syslog(LOG_NOTICE, "There is already an existing pidfile !!!\n" );
	syslog(LOG_NOTICE, "If the baracuda-process isn't already running, please delete /tmp/baracuda_pid\n");
	return(EXIT_FAILURE);
	}
	
FILE *fp = fopen("/tmp/baracuda_pid", "w+");
if(fp == NULL){
	syslog(LOG_NOTICE, "Can't open Pidfile %d\n", pid );
	return EXIT_FAILURE;
	}
	
fwrite( (void *)&pid, sizeof(pid_t), 1, fp );
fclose(fp);

/* open /proc/baracuda/conf to pass the PID */
char proc_pass[50];

fp = fopen("/proc/barracuda/conf", "w+");
if(fp == NULL){
	syslog(LOG_NOTICE, "Can't open /proc/barracuda/conf for pid passing \n" );
	return EXIT_FAILURE;
	}
sprintf( (char *)&proc_pass, "pid=%d", pid);
fwrite( (void *)&proc_pass, strlen(proc_pass), 1, fp );
fclose(fp);
	
/* open /proc/baracuda/conf to setup a connection type */
fp = fopen("/proc/barracuda/conf", "w+");
if(fp == NULL){
	syslog(LOG_NOTICE, "Can't open /proc/barracuda/conf for mode passing \n" );
	return EXIT_FAILURE;
	}

switch( c_mode ){
	case 1 :	sprintf( (char *)&proc_pass, "con=NL");
				fwrite( (void *)&proc_pass, strlen(proc_pass), 1, fp );
				break;
	case 2 :	sprintf( (char *)&proc_pass, "con=IOCTL");
				fwrite( (void *)&proc_pass, strlen(proc_pass), 1, fp );
				break;
	case 3 :	sprintf( (char *)&proc_pass, "con=PROCFS");
				fwrite( (void *)&proc_pass, strlen(proc_pass), 1, fp );
				break;
	default :	return EXIT_FAILURE; 
}
		
fclose(fp);

/* Register a signal-handler which handles init.d stop call */

signal(SIGALRM, alarm_handler);

/* 
 * Register a signal-handler that catches the KILL calls to avoid unexpected
 * Hangups.
 */

signal(SIGTERM, kill_handler);
signal(SIGKILL, kill_handler);

/* open a filepointer for mmaping or copy_to_user */
fd=open("/dev/barracuda", O_RDWR);
if(fd < 0){
	syslog(LOG_NOTICE, "fd opening failed !\n");
	return(-1);
	}

/* Signal that everything is fine */
syslog(LOG_NOTICE, "Daemon-Mode established %d\n", pid );

/* Do something usefull */
switch( c_mode ){
	case 1 :	server_netlink(gen_syndrome);
				break;
	case 2 :	server_ioctl_callback(gen_syndrome);
				break;
	case 3 :	server_procfs(gen_syndrome);
				break;
	default :	putchar('\a'); 
}

/* cleanup section */

/* close the mmaping filepointer */
if(fd >= 0){ close(fd); }

/* delete lock */
remove("/tmp/baracuda_pid");

/* Free the dptr array */
free(global_dptrs);

syslog(LOG_NOTICE, "Mode number was : %d.\n", c_mode);
syslog(LOG_NOTICE, "Baracuda-Deamon terminated, please unload the kernel-module.\n");
return 0;
}



/**
 * This is the handler which catchs the singnal SIGKILL and SIGTERM. This is 
 * necessary, because a simple kill can lead to an undefined state which could 
 * cause the whole system to hang.
 *
 * @param signum	: Signal number
 *
 * @returns			void
 */

void kill_handler(int signum)
{
if( (signum == SIGKILL) || (signum == SIGTERM) ){
	syslog(LOG_NOTICE, "Kill called. Please use <barracuda stop> to shutdown into a secure state.\n");
	syslog(LOG_NOTICE, "This instance of baracuda will remain active!\n");
	/* Restart handler */
	signal (signum, kill_handler);
	}
}



/**
 * This is the signal handler which is called if the userspace frontend is instructed
 * to cleanup the userspace driver instance. The related signal could be called from 
 * baracuda_deamon.c with the command line option <-k>. Alternativly this could
 * be achieved bei calling <kill -SIGALRM [PID]> on the command line. BUT, if you
 * did this in that way you MUST delete the lockfile /tmp/baracuda_pid
 *
 * @param signum	: Signal number
 *
 * @returns			void
 */

void alarm_handler(int signum)
{
FILE *fp;
char flag = 'u';
	
if( signum == SIGALRM){
	/** 
 	 * Check if there are already used MD-devices. They must be unmounted and 
 	 * deregistered before the kernel module can be unloaded. The procfs-entry
	 * under </proc/barracuda/conf> inherits a 'n' if there are no mounted md-devices
	 * and 'u' when devices are mounted.
 	 */	
	
	fp = fopen("/proc/barracuda/conf", "r");
	if(fp == NULL){
		syslog(LOG_NOTICE, "Can't open /proc/barracuda/conf for shutdown checks\n" );
		return;
		}
	
	fread( (void *)&flag, sizeof(char), 1, fp );
	fclose(fp);
	
	if(flag == 'n'){
		syslog(LOG_NOTICE, "Shutdown called!!\n");
	
		/* Terminate all possible loops */
		keep_going = 0;
		syslog(LOG_NOTICE, "SIG Barrier set to zero.\n");
	
		/* Restart handler */
		signal (signum, alarm_handler);
		}
	else{ syslog(LOG_NOTICE, "Shutdown is not possible because ther are alread active RAID-Devices.\n"); }
	}
}



/*SUB_THREADS_________________________________________________________________*/
/**
 * This function is the userspace driver which is implementated with ioctl
 * callback method as the used connection technology.
 *
 * @param gen_syndrome	: 	Function Pointer to one of the RS implementations, which
 *							are located in the raid6*.cu files.
 *
 * @returns				a pointer to the actual syndrome block.
 */

int server_ioctl_callback(syndrome_func gen_syndrome)
{
int fd;
char buffer[] = "flag";

/* syndrome data */
int disks;
size_t bytes;
void **ptrs;
syndrome_container *act_container;

syslog(LOG_NOTICE, "IOCTL-Callback method called.\n");
	
/* Open device file for IOCTL handling */
	
fd = open("/dev/barracuda", O_RDONLY);
if( fd < 0){
	syslog(LOG_NOTICE, "IOCTL handler opening failed\n");
	return -1;
	}
syslog(LOG_NOTICE, "IOCTL handler opened!\n");
	
/* Loop until the deamon is killed */
	
while( keep_going ){
	/* Call the IOCTL */
	ioctl(fd, IOCTL_GETVALUE, &buffer);
	
	/* get actual syndrome pointer */
		
	#ifdef DEBUG_LEVEL_3	
	syslog(LOG_NOTICE, "next : get_act_syndrome_block\n");
	#endif

#ifdef COPY_MARSHALLING
	act_container = copy_act_syndrome_block();
#endif
#ifndef COPY_MARSHALLING
	act_container = get_act_syndrome_block();
#endif
	
	/* Disassemble container */
	disks = act_container->disks;
	bytes = act_container->bytes;
	ptrs  = act_container->ptrs;
	
	/* Pass to the gen syndrome function */
	#ifdef DEBUG_LEVEL_3
	syslog(LOG_NOTICE, "next : gen_syndrome\n");
	#endif
	
	gen_syndrome(disks, bytes, ptrs);
	
	/* unmap everything */
#ifdef COPY_MARSHALLING
	copyback_act_syndrome_block(act_container);
#endif
#ifndef COPY_MARSHALLING
	unget_act_syndrome_block(act_container);
#endif

	}
	
/* Close the opened IOCTL handler */
if(fd >= 0){ close(fd); }

return 0;
}



/**
 * This function is the userspace driver which is implemented with the netlink
 * method as the used connection technology.
 *
 * @param gen_syndrome	: 	Function Pointer to one of the RS implementations, which
 *							are located in the raid6*.cu files.
 *
 * @returns				a pointer to the actual syndrome block.
 */

int server_netlink(syndrome_func gen_syndrome)
{
/* netlink related stuff */
struct sockaddr_nl src_addr;
int sock_fd;
int bind_ret = 0;
	
struct msghdr msg_server;
struct msghdr msg_client;

/* syndrome data */
int disks;
size_t bytes;
void **ptrs;

#ifdef DEBUG_LEVEL_1
	unsigned long date;
#endif


char *buffer = (char *)malloc(sizeof(char)*MAX_PAYLOAD);
syndrome_container *act_container;

syslog(LOG_NOTICE, "Netlink method called.\n");

/* create socket */
sock_fd = socket(PF_NETLINK, SOCK_DGRAM, NETLINK_RS_SERVER);
if(sock_fd < 0){
	syslog(LOG_NOTICE, "Can't create netlink socket.\n");
	exit(0);
	}
	
/* bind socket */
memset( &src_addr, 0, sizeof(src_addr));
src_addr.nl_family = AF_NETLINK;
src_addr.nl_pid = getpid();
src_addr.nl_groups = 0;
	
bind_ret = bind(sock_fd, (struct sockaddr*)&src_addr, sizeof(src_addr));

if( bind_ret < 0 ){
	syslog(LOG_NOTICE, "Can't bind netlink socket.\n");
	exit(0);
	}
	
gen_message_container( &msg_client );
gen_message_container( &msg_server );
	
while( keep_going ){
	/** 
	 * __first init every time__
	 * Netlink communication must always be initialised from the userspace and
	 * is not till then a bidirectional comunication method. For a correct 
	 * working messaging modell, we first have to send an empty packet to the
	 * kernelspace.
	 */
	
	#ifdef DEBUG_LEVEL_1
	syslog(LOG_NOTICE, "next : initial add_payload, sendmsg");
	#endif
		
	add_payload( &msg_server, "init");
	sendmsg(sock_fd, &msg_server, 0);
	
	/**
	 * If there is a syndrome that must be calculated, the kernel sends a message 
	 * to this handler.
	 */
	
	#ifdef DEBUG_LEVEL_1
	syslog(LOG_NOTICE, "next : recvmsg, get_payload");
	#endif
	
	recvmsg(sock_fd, &msg_client, 0);
	get_payload( &msg_client, buffer);
	
	#ifdef DEBUG_LEVEL_1
	memcpy( &date, &buffer, sizeof(unsigned long));
	syslog(LOG_NOTICE, "Adress is : %lu\n", date);
	#endif
		
	#ifdef DEBUG_LEVEL_1	
	syslog(LOG_NOTICE, "next : get_act_syndrome_block\n");
	#endif

	/* get actual syndrome pointer */
#ifdef COPY_MARSHALLING
	act_container = copy_act_syndrome_block();
#endif
#ifndef COPY_MARSHALLING
	act_container = get_act_syndrome_block();
#endif
	
	/* Disassemble container */
	disks = act_container->disks;
	bytes = act_container->bytes;
	ptrs  = act_container->ptrs;
	
	/* Pass to the gen syndrome function */
	#ifdef DEBUG_LEVEL_1
	syslog(LOG_NOTICE, "next : gen_syndrome\n");
	#endif
	
	gen_syndrome(disks, bytes, ptrs);
	
	/* unmap everything */
#ifdef COPY_MARSHALLING
	copyback_act_syndrome_block(act_container);
#endif
#ifndef COPY_MARSHALLING
	unget_act_syndrome_block(act_container);
#endif
	
	/* Acknowledge that all calculations are done */
	#ifdef DEBUG_LEVEL_1
	syslog(LOG_NOTICE, "next : add_payload\n");
	#endif
	
	add_payload( &msg_server, buffer);
	
	#ifdef DEBUG_LEVEL_1
	syslog(LOG_NOTICE, "next : sendmsg\n");
	#endif
		
	sendmsg(sock_fd, &msg_server, 0);
	}
	
close(sock_fd);

destroy_message_container( &msg_server);
destroy_message_container( &msg_client);
free(buffer);
	
return 0;
}



/**
 * This function is the userspace driver which is implementated with the procfs
 * method as the used connection technology.
 *
 * @param gen_syndrome	: 	Function Pointer to one of the RS implementations, which
 *							are located in the raid6*.cu files.
 *
 * @returns				a pointer to the actual syndrome block.
 */

int server_procfs(syndrome_func gen_syndrome)
{
FILE *fd;
syndrome_container *act_container;
char buffer[sizeof(unsigned long)+2];
#ifdef DEBUG_LEVEL_1
	double time;
	unsigned long date;
#endif

/*
int i,j;
char *tmp;
*/
	
/* syndrome data */
int disks;
size_t bytes;
void **ptrs;
	
syslog(LOG_NOTICE, "Procfs method called.\n");
	
fd = fopen("/proc/barracuda/stub", "w+");
if( fd < 0){
	syslog(LOG_NOTICE, "Proc stub open failed!!\n");
	return -1;
	}
syslog(LOG_NOTICE, "Procfs handler opened.\n");

while( keep_going ){
	fread( &buffer, sizeof(char), sizeof(unsigned long)+2, fd );
	
	#ifdef DEBUG_LEVEL_1
	memcpy( &date, &buffer, sizeof(unsigned long));
	syslog(LOG_NOTICE, "Adress is : %lu\n", date);
	#endif
	
	/* get actual syndrome pointer */
#ifdef COPY_MARSHALLING
	act_container = copy_act_syndrome_block();
#endif
#ifndef COPY_MARSHALLING
	act_container = get_act_syndrome_block();
#endif
	
	/* deassemble container */
	disks = act_container->disks;
	bytes = act_container->bytes;
	ptrs  = act_container->ptrs;
	
	/*
	for(i=0; i<disks; i++){
		tmp = (char *)ptrs[i];
		syslog(LOG_NOTICE, "%d\n", tmp[0]);
		}
	*/
	
	#ifdef DEBUG_LEVEL_1
	time = gtd_second();
	#endif
	/* pass to the gen syndrome function */
	gen_syndrome(disks, bytes, ptrs);
	#ifdef DEBUG_LEVEL_1
	time = gtd_second()-time;
	syslog(LOG_NOTICE, "TIME for gensyn() : %f milli\n", time*1000);
	#endif
	
	/* unmap everything */
#ifdef COPY_MARSHALLING
	copyback_act_syndrome_block(act_container);
#endif
#ifndef COPY_MARSHALLING
	unget_act_syndrome_block(act_container);
#endif
	
	/* acknowledge that all calculations are done */
	fwrite( &buffer, sizeof(char), sizeof(unsigned long)+2, fd );
	}
	
/* Close the file-pointer */
if(fd >= 0){ fclose(fd); }
	
return 0;
}



/*HELPER_FUNCTIONS____________________________________________________________*/
/**
 * Copy actual syndrome container from kernelspace via copy_to_user
 *
 * @returns	 syndrome_container * : A pointer to the actual syndrome
 *									which should be calculated.
 */

syndrome_container *copy_act_syndrome_block( )
{
#ifdef DEBUG_LEVEL_1
double time;
double total_time = 0;
#endif

int i;
syndrome_container *ret;
unsigned int pagesizen = getpagesize();

int disks;
size_t bytes;
void **dptrs = global_dptrs;

/* map the marshalling struct */
if(smc_flag == 0){
	ret_global = (syndrome_container *)mmap(0, sizeof(syndrome_container), PROT_READ, MAP_SHARED, fd, 0);
	if(ret_global == MAP_FAILED){
		perror("MMAPing marshalling struct failed !\n");
		return NULL;
		}
	smc_flag = 1;
	}
	
disks = ret_global->disks;
bytes = ret_global->bytes;
	
#ifdef DEBUG_LEVEL_1
	time = gtd_second();
#endif

/* malloc all data buffers */
for(i=0; i<disks; i++){
	dptrs[i] = (void *)malloc(bytes);
	}
	
#ifdef DEBUG_LEVEL_1
	time = gtd_second()-time;
	total_time = total_time + time;
	syslog(LOG_NOTICE, ">>> copy dpointer : %f milli\n", time*1000);
#endif

/* copy the stuff from the kernelspace */
for( i=0; i<disks-2; i++){
	pread(fd, dptrs[i], bytes, i);
	}
	
#ifdef DEBUG_LEVEL_1
	time = gtd_second();
#endif
	
/* malloc a syndrome container that resides at the userspace */
ret = (syndrome_container *)malloc( sizeof(syndrome_container) );

/* put all arguments into the marshalling struct */
ret->disks = disks;
ret->bytes = bytes;
ret->ptrs  = dptrs;

#ifdef DEBUG_LEVEL_1
	time = gtd_second()-time;
	total_time = total_time + time;
	syslog(LOG_NOTICE, ">>> put struct : %f milli\n", time*1000);
	syslog(LOG_NOTICE, "TOTAL mmap() -> %f milli\n", total_time*1000);
#endif
	
return ret;
}



/**
 * Copy back actual syndrome container to kernelspace via copy_to_user
 *
 * @param    smc : A pointer to the actual syndrome which should be calculated.
 *
 * @returns	 void
 */

void copyback_act_syndrome_block( syndrome_container *smc )
{
int disks		= smc->disks;
size_t bytes	= smc->bytes;
void **dptrs	= smc->ptrs;
	
int i;
	
/* copy all checksums back to the kernelspace */
pwrite(fd, dptrs[disks-2], bytes, disks-2);
pwrite(fd, dptrs[disks-1], bytes, disks-1);
	
/* free all buffers */
for(i=0; i<disks; i++){
	free(smc->ptrs[i]);
	}
free(smc);
}



/**
 * Get the actual syndrome container from the kernelspace
 *
 *
 * @returns	 syndrome_container * : A pointer to the actual syndrome
 *									which should be calculated.
 */

syndrome_container *get_act_syndrome_block()
{
#ifdef DEBUG_LEVEL_1
	double time;
	double total_time = 0;
#endif
int i;
syndrome_container *ret;
unsigned int pagesizen = getpagesize();

int disks;
size_t bytes;
void **dptrs = global_dptrs;
	
#ifdef DEBUG_LEVEL_1
	time = gtd_second();
#endif
/* map the marshalling struct */
if(smc_flag == 0){
	ret_global = (syndrome_container *)mmap(0, sizeof(syndrome_container), PROT_READ, MAP_SHARED, fd, 0);
	if(ret_global == MAP_FAILED){
		perror("MMAPing marshalling struct failed !\n");
		return NULL;
		}
	smc_flag = 1;
	}
#ifdef DEBUG_LEVEL_1
	time = gtd_second()-time;
	total_time = total_time + time;
	syslog(LOG_NOTICE, ">>> map marshal : %f milli\n", time*1000);
#endif

/* allocate pointers for the disks array */
disks = ret_global->disks;
bytes = ret_global->bytes;
		
/* map every disk pointer individually */
#ifdef DEBUG_LEVEL_1
	time = gtd_second();
#endif
for(i=1; i <= disks; i++){
	dptrs[i-1] = (void *)mmap(0, bytes, PROT_WRITE, MAP_SHARED, fd, i*pagesizen);
	
	if(dptrs[i-1] == MAP_FAILED){
		perror("MMAPing disk data failed !\n");
		return NULL;
		}
	}
#ifdef DEBUG_LEVEL_1
	time = gtd_second()-time;
	total_time = total_time + time;
	syslog(LOG_NOTICE, ">>> map dpointer : %f milli\n", time*1000);
#endif

#ifdef DEBUG_LEVEL_1
	time = gtd_second();
#endif
	
/* malloc a syndrome container that resides at the userspace */
ret = (syndrome_container *)malloc( sizeof(syndrome_container) );

/* put all arguments into the marshalling struct */
ret->disks = disks;
ret->bytes = bytes;
ret->ptrs  = dptrs;

#ifdef DEBUG_LEVEL_1
	time = gtd_second()-time;
	total_time = total_time + time;
	syslog(LOG_NOTICE, ">>> put struct : %f milli\n", time*1000);
	syslog(LOG_NOTICE, "TOTAL mmap() -> %f milli\n", total_time*1000);
#endif

return ret;
}



/**
 * Unmap the actual syndrome container via munmap
 *
 * @param *smc		: syndrome container to unmap
 *
 * @returns			void
 */

void unget_act_syndrome_block( syndrome_container *smc )
{
int i;
int disks = smc->disks;
size_t bytes = smc->bytes;

#ifdef DEBUG_LEVEL_1
	double time;
	double total_time = 0;
#endif

#ifdef DEBUG_LEVEL_1
	time = gtd_second();
#endif

/* First unmap all datapointer stuff */
for(i=0; i < disks; i++){
	munmap( smc->ptrs[i], bytes );
	}

#ifdef DEBUG_LEVEL_1
	time = gtd_second()-time;
	total_time = total_time + time;
	syslog(LOG_NOTICE, ">>> unmap dpointer : %f milli\n", time*1000);
#endif

#ifdef DEBUG_LEVEL_1
	time = gtd_second();
#endif

/* At last, free the marshalling struct */
free(smc);

#ifdef DEBUG_LEVEL_1
	time = gtd_second()-time;
	total_time = total_time + time;
	syslog(LOG_NOTICE, ">>> free smc : %f milli\n", time*1000);
	syslog(LOG_NOTICE, "TOTAL munmap() -> %f milli\n", total_time*1000);
#endif
}



/**
 * Generates the packet structure for the netlink protocoll.
 *
 * @param msg		: packet datastructure
 *
 * @returns			void
 */

void gen_message_container(struct msghdr *msg)
{
/* init payload package */
struct nlmsghdr *nlh = NULL;	
nlh = (struct nlmsghdr *)malloc(NLMSG_SPACE(MAX_PAYLOAD));
memset(nlh, 0, NLMSG_SPACE(MAX_PAYLOAD));

nlh->nlmsg_len   = NLMSG_SPACE(MAX_PAYLOAD);
nlh->nlmsg_pid   = getpid();
nlh->nlmsg_flags = NLM_F_REQUEST| NLM_F_ECHO;
nlh->nlmsg_type  = NL_COMMAND;
	
/* init iovec struct */
struct iovec *iov;
iov = (struct iovec *)malloc(sizeof(struct iovec));

iov->iov_base = (void *)nlh;
iov->iov_len  = nlh->nlmsg_len;

/* init struct for the destination adress */
struct sockaddr_nl *dst_addr;
dst_addr = (struct sockaddr_nl *)malloc(sizeof(struct sockaddr_nl));
memset( dst_addr, 0, sizeof(dst_addr));

dst_addr->nl_family = AF_NETLINK;
dst_addr->nl_pid = 0;
dst_addr->nl_groups = 0;

/* init struct msghdr */
memset( msg, 0, sizeof(struct msghdr));

msg->msg_name    = dst_addr;
msg->msg_namelen = sizeof(struct sockaddr_nl);
msg->msg_iov     = iov;
msg->msg_iovlen  = 1;
}



/**
 * Deallocates the netlink packet-header
 *
 * @param msg		: packet datastructure
 *
 * @returns			void
 */

void destroy_message_container(struct msghdr *msg)
{
free(msg->msg_iov->iov_base);
free(msg->msg_iov);
free(msg->msg_name);
}



/**
 * Save the payload to a Netlink-Package for sending
 *
 * @param *msg					: Target function pointer
 * @param *payload				: Array pointer to the availaible implementations
 *
 * @returns			void
 */

void add_payload(struct msghdr *msg, char *payload )
{
struct nlmsghdr *nlh;

#ifdef NOCUDA
	nlh = msg->msg_iov->iov_base;
#else
	nlh = (nlmsghdr *)msg->msg_iov->iov_base;
#endif

strcpy( (char *)NLMSG_DATA(nlh), payload );	
}



/**
 * Get the payload from a received Netlink-Package
 *
 * @param *msg					: Target function pointer
 * @param *payload				: Array pointer to the availaible implementations
 *
 * @returns			void
 */

void get_payload(struct msghdr *msg, char *payload)
{
struct nlmsghdr *nlh;

#ifdef NOCUDA
	nlh = msg->msg_iov->iov_base;
#else
	nlh = (nlmsghdr *)msg->msg_iov->iov_base;
#endif

strcpy(payload, (char *)NLMSG_DATA(nlh));
}
