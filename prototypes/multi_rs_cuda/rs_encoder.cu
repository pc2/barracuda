/**
 * \file
 * \brief	Multi error correcting rs-encoder prototype
 *
 * @author	Dominic Eschweiler weiler@upb.de
 *
 * Status	: ALPHA \n
 * Date of creation : 16.12.2008
 *
 */

/*****************************************************************
 *
 * Barracuda is a experimental microdriver extension to the 
 * linux-kernel that is able to outsource common functions to
 * the userspace. It was intensionally designed to accelerate
 * CPU-intensive Tasks on a GPU.
 *
 * Copyright (C) 2009 Dominic Eschweiler
 *
 * This program is free software; you can redistribute it and/or 
 * modify it under the terms of the GNU General Public License as 
 * published by the Free Software Foundation; either only GPLv2 - 
 * version 2 of the License.
 *
 * This program is distributed in the hope that it will be useful, 
 * but WITHOUT ANY WARRANTY; without even the implied warranty of 
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the 
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public 
 * License along with this program; 
 * if not, see <http://www.gnu.org/licenses/>.
 *
 *****************************************************************/

#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <string.h>
#include <dirent.h>
#include <sys/stat.h>
#include <sys/file.h>
#include <sys/types.h>
#include <time.h>
#include <sys/time.h>
#include <math.h>
#include <sched.h>
#include <signal.h>
#include <sys/wait.h>
#include <syslog.h>

#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

//#define GEN
#define DISKS 9
#define CHECK_SYMBOLS 3
//#define BYTES 4096
#define BYTES 524288
//#define BYTES 4128


#define NW (1 << w)
//#define DEBUG_MULT_RS
#define CUDA_COPY
#define KERNEL_EXEC

//#define DMA_BLOCKSIZE 4096
//#define THREAD_BLOCKSIZE 64

#define DMA_BLOCKSIZE 524288
#define THREAD_BLOCKSIZE 256

//#define DMA_BLOCKSIZE 256
//#define THREAD_BLOCKSIZE 16


extern void multi_rs_cuda_gen_syndrome(int disks, size_t bytes, void **ptrs);

extern void multi_rs_soft_gen_syndrome(int disks, size_t bytes, void **ptrs);
inline unsigned char mult_gf(unsigned char a, unsigned char b);

__global__ void rs_kernel( unsigned char *DEVICE_DP, unsigned char *DEVICE_PQ, int disks);
__device__ inline unsigned char mult_gf_shader(unsigned char a, unsigned char b, unsigned char gflog[], unsigned char gfilog[]);
static void inline get_card_mem(void);

int setup_tables(void);
double gtd_second(void);

static unsigned char gflog[] = {
0, 0, 1, 25, 2, 50, 26, 198, 3, 223, 51, 238, 27, 104, 199, 75, 4, 100, 224, 14,
52, 141, 239, 129, 28, 193, 105, 248, 200, 8, 76, 113, 5, 138, 101, 47, 225, 36,
15, 33, 53, 147, 142, 218, 240, 18, 130, 69, 29, 181, 194, 125, 106, 39, 249, 185,
201, 154, 9, 120, 77, 228, 114, 166, 6, 191, 139, 98, 102, 221, 48, 253, 226, 152,
37, 179, 16, 145, 34, 136, 54, 208, 148, 206, 143, 150, 219, 189, 241, 210, 19,
92, 131, 56, 70, 64, 30, 66, 182, 163, 195, 72, 126, 110, 107, 58, 40, 84, 250,
133, 186, 61, 202, 94, 155, 159, 10, 21, 121, 43, 78, 212, 229, 172, 115, 243, 167,
87, 7, 112, 192, 247, 140, 128, 99, 13, 103, 74, 222, 237, 49, 197, 254, 24, 227,
165, 153, 119, 38, 184, 180, 124, 17, 68, 146, 217, 35, 32, 137, 46, 55, 63, 209,
91, 149, 188, 207, 205, 144, 135, 151, 178, 220, 252, 190, 97, 242, 86, 211, 171,
20, 42, 93, 158, 132, 60, 57, 83, 71, 109, 65, 162, 31, 45, 67, 216, 183, 123, 164,
118, 196, 23, 73, 236, 127, 12, 111, 246, 108, 161, 59, 82, 41, 157, 85, 170, 251,
96, 134, 177, 187, 204, 62, 90, 203, 89, 95, 176, 156, 169, 160, 81, 11, 245, 22,
235, 122, 117, 44, 215, 79, 174, 213, 233, 230, 231, 173, 232, 116, 214, 244, 234,
168, 80, 88};

static unsigned char gfilog[] = {
1, 2, 4, 8, 16, 32, 64, 128, 29, 58, 116, 232, 205, 135, 19, 38, 76, 152, 45, 90,
180, 117, 234, 201, 143, 3, 6, 12, 24, 48, 96, 192, 157, 39, 78, 156, 37, 74, 148,
53, 106, 212, 181, 119, 238, 193, 159, 35, 70, 140, 5, 10, 20, 40, 80, 160, 93,
186, 105, 210, 185, 111, 222, 161, 95, 190, 97, 194, 153, 47, 94, 188, 101, 202,
137, 15, 30, 60, 120, 240, 253, 231, 211, 187, 107, 214, 177, 127, 254, 225, 223,
163, 91, 182, 113, 226, 217, 175, 67, 134, 17, 34, 68, 136, 13, 26, 52, 104, 208,
189, 103, 206, 129, 31, 62, 124, 248, 237, 199, 147, 59, 118, 236, 197, 151, 51,
102, 204, 133, 23, 46, 92, 184, 109, 218, 169, 79, 158, 33, 66, 132, 21, 42, 84,
168, 77, 154, 41, 82, 164, 85, 170, 73, 146, 57, 114, 228, 213, 183, 115, 230, 209,
191, 99, 198, 145, 63, 126, 252, 229, 215, 179, 123, 246, 241, 255, 227, 219, 171,
75, 150, 49, 98, 196, 149, 55, 110, 220, 165, 87, 174, 65, 130, 25, 50, 100, 200,
141, 7, 14, 28, 56, 112, 224, 221, 167, 83, 166, 81, 162, 89, 178, 121, 242, 249,
239, 195, 155, 43, 86, 172, 69, 138, 9, 18, 36, 72, 144, 61, 122, 244, 245, 247,
243, 251, 235, 203, 139, 11, 22, 44, 88, 176, 125, 250, 233, 207, 131, 27, 54,
108, 216, 173, 71, 142
};

unsigned short *gflog2, *gfilog2;


/**
 * Main function of control
 *
 * @returns			int
 */

int main()
{
#ifdef GEN
setup_tables();
#endif

#ifndef GEN
int i,j ;
void **dptrs;
unsigned char* tmp;
	
dptrs=(void **)malloc( DISKS*sizeof(void *) );
for(i=0; i < DISKS; i++){
	dptrs[i] = malloc(BYTES);
	}
	
for ( i=0 ; i < DISKS ; i++ ){
	memset(dptrs[i], i, BYTES);
	tmp = (unsigned char *)dptrs[i];
	for ( j=0 ; j < BYTES ; j++ ){
		//printf("%u ", tmp[j]);
		}
	//printf("\n\n");
	}
	
printf("____\n");

//______________________________________________________________________________

double timer, tmp_timer;
	
timer = 0; j = 0;
while(timer < 2){
	tmp_timer = gtd_second();
		multi_rs_soft_gen_syndrome(DISKS, BYTES, dptrs);
	tmp_timer = gtd_second() - tmp_timer;
	timer = timer + tmp_timer;
	j++;
	}
printf("%d ; %u\n", 1, (unsigned long)((BYTES*j)/timer) );
	
timer = 0; j = 0;
while(timer < 2){
	tmp_timer = gtd_second();
		multi_rs_cuda_gen_syndrome(DISKS, BYTES, dptrs);
	tmp_timer = gtd_second() - tmp_timer;
	timer = timer + tmp_timer;
	j++;
	}
printf("%d ; %u\n", 2, (unsigned long)((BYTES*j)/timer) );
	
//multi_rs_soft_gen_syndrome(DISKS, BYTES, dptrs);
//multi_rs_cuda_gen_syndrome(DISKS, BYTES, dptrs);

//______________________________________________________________________________
	
for ( i=0 ; i < DISKS ; i++ ){
	tmp = (unsigned char *)dptrs[i];
	for ( j=0 ; j < BYTES ; j++ ){
		//printf("%u ", tmp[j]);
		}
	//printf("\n\n");
	}

for(i = 0; i < DISKS; i++){
	free(dptrs[i]);
	}
free(dptrs);
#endif
	
return 0;
}

//______________________________________________________________________________

/**
 * This is a multi failure correcting version of gen_syndrome which runs entirely
 * on the cpu.
 *
 * @param disks		: # of disks
 * @param bytes		: # number of bytes
 * @param **ptrs	: processing data
 *
 * @returns			void
 */

extern void multi_rs_soft_gen_syndrome(int disks, size_t bytes, void **ptrs)
{
unsigned char matrix_pos_y;
unsigned char matrix_pos_x;
#ifdef DEBUG_MULT_RS
unsigned char tmp;
#endif
	
unsigned char **dptrs = (unsigned char **)ptrs;
	
int i;
int j;
int d;
	
int high_disk = (disks-CHECK_SYMBOLS);
	
for(i=0; i<bytes; i++){
	matrix_pos_y = 0;
	#ifdef DEBUG_MULT_RS
	printf("byte %d\n", i);
	#endif
	
	for(j=high_disk; j<disks; j++){
		matrix_pos_y++;
		matrix_pos_x = matrix_pos_y;
		#ifdef DEBUG_MULT_RS
		printf("%u", matrix_pos_y);
		#endif
		
		dptrs[j][i] = 0;
		for(d=0; d<high_disk; d++){
			dptrs[j][i] ^= mult_gf(matrix_pos_x, dptrs[d][i]);
			#ifdef DEBUG_MULT_RS
			tmp = mult_gf(matrix_pos_x, dptrs[d][i]);
			printf("[(%u*%u) = %u] +", matrix_pos_x, dptrs[d][i], tmp);
			#endif
			matrix_pos_x  = mult_gf(matrix_pos_x, matrix_pos_y);
			}
		#ifdef DEBUG_MULT_RS
		printf("\b = %u\n", dptrs[j][i]);
		#endif
		}
	#ifdef DEBUG_MULT_RS
	printf("\n");
	#endif
	}
}



/**
 * This function implements multiplication on an GF(2) with lookup tables
 *
 * @param a		: first operand
 * @param b		: second operand
 *
 * @returns			result of the GF(2) multiplication
 */

inline unsigned char mult_gf(unsigned char a, unsigned char b)
{
unsigned char sum_log;
int w = 8;
	
if(a==0 || b==0){return 0;}
	
sum_log = gflog[a] + gflog[b];
  
if(sum_log >= NW-1){sum_log -= NW-1;}
 
return gfilog[sum_log];
}

//______________________________________________________________________________

static unsigned char *DEVICE_DP;
static unsigned char *DEVICE_CS;
static int mem_tag = 0;



/**
 * This is a multi failure correcting version of gen_syndrome which runs entirely
 * on the gpu. This function is the body which uploads and executes the compute
 * kernel.
 *
 * @param disks		: # of disks
 * @param bytes		: # number of bytes
 * @param **ptrs	: processing data
 *
 * @returns			void
 */

extern void multi_rs_cuda_gen_syndrome(int disks, size_t bytes, void **ptrs)
{
int i, j;
dim3 dimBlock;
dim3 dimGrid;
	
unsigned char **dptrs = (unsigned char **)ptrs;
unsigned long runs  = floor(bytes/DMA_BLOCKSIZE);
unsigned long carry = bytes - (runs * DMA_BLOCKSIZE);
	
get_card_mem();
	
#ifdef DEBUG_MULT_RS
printf("block_x : %d, dimgrid_x : %d, runs : %d\n", dimBlock.x, dimGrid.x, runs);
cudaError_t error_t;
error_t = cudaGetLastError();
printf("cuda_copy : %s\n", cudaGetErrorString(error_t) );
#endif

dimBlock.x=THREAD_BLOCKSIZE;
dimBlock.y=1;
dimBlock.z=1;
dimGrid.x=(DMA_BLOCKSIZE/(dimBlock.x*8));
dimGrid.y=1;
		
for(j=0; j<runs; j++){	
	#ifdef CUDA_COPY
	for(i=0; i<disks-CHECK_SYMBOLS; i++){		
		hipMemcpy( &DEVICE_DP[i*DMA_BLOCKSIZE], 
				   	&dptrs[i][j*DMA_BLOCKSIZE],
					DMA_BLOCKSIZE, 
					hipMemcpyHostToDevice);
		}
	#endif
	
	#ifdef KERNEL_EXEC
	rs_kernel<<<dimGrid, dimBlock>>>( DEVICE_DP, DEVICE_CS, disks);
	#endif
	
	#ifdef DEBUG_MULT_RS
		error_t = cudaGetLastError();
		printf("kernel exec : %s\n", cudaGetErrorString(error_t) );
	#endif
	
	#ifdef CUDA_COPY
	for(i=0; i<CHECK_SYMBOLS; i++){		
		hipMemcpy( &dptrs[(disks-CHECK_SYMBOLS)+i][j*DMA_BLOCKSIZE],
				    &DEVICE_CS[i*DMA_BLOCKSIZE],
					DMA_BLOCKSIZE, 
					hipMemcpyDeviceToHost);
		}
	#endif
	
	#ifdef DEBUG_MULT_RS
		error_t = cudaGetLastError();
		printf("cuda_copy_back : %s\n", cudaGetErrorString(error_t) );
	#endif
	}
	
dimBlock.x=THREAD_BLOCKSIZE;
dimBlock.y=1;
dimBlock.z=1;
dimGrid.x=ceil(carry/(dimBlock.x*8));
dimGrid.y=1;
	
if(carry > 0){
	#ifdef CUDA_COPY
	for(i=0; i<disks-CHECK_SYMBOLS; i++){		
		hipMemcpy( &DEVICE_DP[i*DMA_BLOCKSIZE], 
				   	&dptrs[i][j*carry],
					DMA_BLOCKSIZE, 
					hipMemcpyHostToDevice);
		}
	#endif
	
	#ifdef KERNEL_EXEC
	rs_kernel<<<dimGrid, dimBlock>>>( DEVICE_DP, DEVICE_CS, disks);
	#endif
	
	#ifdef DEBUG_MULT_RS
		error_t = cudaGetLastError();
		printf("kernel exec : %s\n", cudaGetErrorString(error_t) );
	#endif
	
	#ifdef CUDA_COPY
	for(i=0; i<CHECK_SYMBOLS; i++){		
		hipMemcpy( &dptrs[(disks-CHECK_SYMBOLS)+i][j*carry],
				    &DEVICE_CS[i*DMA_BLOCKSIZE],
					DMA_BLOCKSIZE, 
					hipMemcpyDeviceToHost);
		}
	#endif
	
	#ifdef DEBUG_MULT_RS
		error_t = cudaGetLastError();
		printf("cuda_copy_back : %s\n", cudaGetErrorString(error_t) );
	#endif
	}

}


/**
 * This is the rs kernel function.
 *
 * @param *DEVICE_DP : linearized datapointer
 * @param *DEVICE_PQ : linearized checksum pointer
 * @param disks		 : number of drives
 *
 * @returns			void
 */

__global__ void rs_kernel( unsigned char *DEVICE_DP, unsigned char *DEVICE_PQ, int disks)
{
int bx = blockIdx.x;
int tx = threadIdx.x;
int th = ((bx*THREAD_BLOCKSIZE)+tx);

// logarithm
__shared__ unsigned char k_gflog[256];
__shared__ unsigned char k_gfilog[256];

if(tx == 0){
k_gflog[0] = 0;     k_gflog[1] = 0;     k_gflog[2] = 1;     k_gflog[3] = 25;    
k_gflog[4] = 2;     k_gflog[5] = 50;    k_gflog[6] = 26;    k_gflog[7] = 198;   
k_gflog[8] = 3;     k_gflog[9] = 223;   k_gflog[10] = 51;   k_gflog[11] = 238;  
k_gflog[12] = 27;   k_gflog[13] = 104;  k_gflog[14] = 199;  k_gflog[15] = 75;   
k_gflog[16] = 4;    k_gflog[17] = 100;  k_gflog[18] = 224;  k_gflog[19] = 14;
k_gflog[20] = 52;   k_gflog[21] = 141;  k_gflog[22] = 239;  k_gflog[23] = 129;  
k_gflog[24] = 28;   k_gflog[25] = 193;  k_gflog[26] = 105;  k_gflog[27] = 248;  
k_gflog[28] = 200;  k_gflog[29] = 8;    k_gflog[30] = 76;   k_gflog[31] = 113;
	
// inverse logarithm
k_gfilog[0] = 1;    k_gfilog[1] = 2;    k_gfilog[2] = 4;    k_gfilog[3] = 8;
k_gfilog[4] = 16;   k_gfilog[5] = 32;   k_gfilog[6] = 64;   k_gfilog[7] = 128;  
k_gfilog[8] = 29;   k_gfilog[9] = 58;   k_gfilog[10] = 116; k_gfilog[11] = 232; 
k_gfilog[12] = 205; k_gfilog[13] = 135; k_gfilog[14] = 19;  k_gfilog[15] = 38;  
k_gfilog[16] = 76;  k_gfilog[17] = 152; k_gfilog[18] = 45;  k_gfilog[19] = 90;
k_gfilog[20] = 180; k_gfilog[21] = 117; k_gfilog[22] = 234; k_gfilog[23] = 201;
k_gfilog[24] = 143; k_gfilog[25] = 3;   k_gfilog[26] = 6;   k_gfilog[27] = 12;  
k_gfilog[28] = 24;  k_gfilog[29] = 48;  k_gfilog[30] = 96;  k_gfilog[31] = 192; 
}

if(tx == 1){
k_gflog[32] = 5;    k_gflog[33] = 138;  k_gflog[34] = 101;  k_gflog[35] = 47;   
k_gflog[36] = 225;  k_gflog[37] = 36;   k_gflog[38] = 15;   k_gflog[39] = 33;
k_gflog[40] = 53;   k_gflog[41] = 147;  k_gflog[42] = 142;  k_gflog[43] = 218;  
k_gflog[44] = 240;  k_gflog[45] = 18;   k_gflog[46] = 130;  k_gflog[47] = 69;
k_gflog[48] = 29;   k_gflog[49] = 181;  k_gflog[50] = 194;  k_gflog[51] = 125;  
k_gflog[52] = 106;  k_gflog[53] = 39;   k_gflog[54] = 249;  k_gflog[55] = 185;  
k_gflog[56] = 201;  k_gflog[57] = 154;  k_gflog[58] = 9;    k_gflog[59] = 120;
k_gflog[60] = 77;   k_gflog[61] = 228;  k_gflog[62] = 114;  k_gflog[63] = 166;
	
// inverse logarithm
k_gfilog[32] = 157; k_gfilog[33] = 39;  k_gfilog[34] = 78;  k_gfilog[35] = 156; 
k_gfilog[36] = 37;  k_gfilog[37] = 74;  k_gfilog[38] = 148; k_gfilog[39] = 53;
k_gfilog[40] = 106; k_gfilog[41] = 212; k_gfilog[42] = 181; k_gfilog[43] = 119; 
k_gfilog[44] = 238; k_gfilog[45] = 193; k_gfilog[46] = 159; k_gfilog[47] = 35;  
k_gfilog[48] = 70;  k_gfilog[49] = 140; k_gfilog[50] = 5;   k_gfilog[51] = 10;  
k_gfilog[52] = 20;  k_gfilog[53] = 40;  k_gfilog[54] = 80;  k_gfilog[55] = 160; 
k_gfilog[56] = 93;  k_gfilog[57] = 186; k_gfilog[58] = 105; k_gfilog[59] = 210;
k_gfilog[60] = 185; k_gfilog[61] = 111; k_gfilog[62] = 222; k_gfilog[63] = 161;
}

if(tx == 2){
k_gflog[64] = 6;    k_gflog[65] = 191;  k_gflog[66] = 139;  k_gflog[67] = 98;   
k_gflog[68] = 102;  k_gflog[69] = 221;  k_gflog[70] = 48;   k_gflog[71] = 253;  
k_gflog[72] = 226;  k_gflog[73] = 152;  k_gflog[74] = 37;   k_gflog[75] = 179;  
k_gflog[76] = 16;   k_gflog[77] = 145;  k_gflog[78] = 34;   k_gflog[79] = 136;
k_gflog[80] = 54;   k_gflog[81] = 208;  k_gflog[82] = 148;  k_gflog[83] = 206;  
k_gflog[84] = 143;  k_gflog[85] = 150;  k_gflog[86] = 219;  k_gflog[87] = 189;
k_gflog[88] = 241;  k_gflog[89] = 210;  k_gflog[90] = 19;   k_gflog[91] = 92;
k_gflog[92] = 131;  k_gflog[93] = 56;   k_gflog[94] = 70;   k_gflog[95] = 64;

// inverse logarithm
k_gfilog[64] = 95;  k_gfilog[65] = 190; k_gfilog[66] = 97;  k_gfilog[67] = 194;
k_gfilog[68] = 153; k_gfilog[69] = 47;  k_gfilog[70] = 94;  k_gfilog[71] = 188; 
k_gfilog[72] = 101; k_gfilog[73] = 202; k_gfilog[74] = 137; k_gfilog[75] = 15;
k_gfilog[76] = 30;  k_gfilog[77] = 60;  k_gfilog[78] = 120; k_gfilog[79] = 240;
k_gfilog[80] = 253; k_gfilog[81] = 231; k_gfilog[82] = 211; k_gfilog[83] = 187; 
k_gfilog[84] = 107; k_gfilog[85] = 214; k_gfilog[86] = 177; k_gfilog[87] = 127; 
k_gfilog[88] = 254; k_gfilog[89] = 225; k_gfilog[90] = 223; k_gfilog[91] = 163; 
k_gfilog[92] = 91;  k_gfilog[93] = 182; k_gfilog[94] = 113; k_gfilog[95] = 226;
}

if(tx == 3){
k_gflog[96] = 30;   k_gflog[97] = 66;   k_gflog[98] = 182;  k_gflog[99] = 163;
k_gflog[100] = 195; k_gflog[101] = 72;  k_gflog[102] = 126; k_gflog[103] = 110; 
k_gflog[104] = 107; k_gflog[105] = 58;  k_gflog[106] = 40;  k_gflog[107] = 84;  
k_gflog[108] = 250; k_gflog[109] = 133; k_gflog[110] = 186; k_gflog[111] = 61;  
k_gflog[112] = 202; k_gflog[113] = 94;  k_gflog[114] = 155; k_gflog[115] = 159; 
k_gflog[116] = 10;  k_gflog[117] = 21;  k_gflog[118] = 121; k_gflog[119] = 43;
k_gflog[120] = 78;  k_gflog[121] = 212; k_gflog[122] = 229; k_gflog[123] = 172;
k_gflog[124] = 115; k_gflog[125] = 243; k_gflog[126] = 167; k_gflog[127] = 87;

// inverse logarithm
k_gfilog[96] = 217; k_gfilog[97] = 175; k_gfilog[98] = 67;  k_gfilog[99] = 134;
k_gfilog[100] = 17; k_gfilog[101] = 34; k_gfilog[102] = 68; k_gfilog[103] = 136;
k_gfilog[104] = 13; k_gfilog[105] = 26; k_gfilog[106] = 52; k_gfilog[107] = 104;
k_gfilog[108] = 208;k_gfilog[109] = 189;k_gfilog[110] = 103;k_gfilog[111] = 206;
k_gfilog[112] = 129;k_gfilog[113] = 31; k_gfilog[114] = 62; k_gfilog[115] = 124;
k_gfilog[116] = 248;k_gfilog[117] = 237;k_gfilog[118] = 199;k_gfilog[119] = 147;
k_gfilog[120] = 59; k_gfilog[121] = 118;k_gfilog[122] = 236;k_gfilog[123] = 197;
k_gfilog[124] = 151;k_gfilog[125] = 51; k_gfilog[126] = 102;k_gfilog[127] = 204;
}

if(tx == 4){
k_gflog[128] = 7;   k_gflog[129] = 112; k_gflog[130] = 192; k_gflog[131] = 247;
k_gflog[132] = 140; k_gflog[133] = 128; k_gflog[134] = 99;  k_gflog[135] = 13;  
k_gflog[136] = 103; k_gflog[137] = 74;  k_gflog[138] = 222; k_gflog[139] = 237; 
k_gflog[140] = 49;  k_gflog[141] = 197; k_gflog[142] = 254; k_gflog[143] = 24;  
k_gflog[144] = 227; k_gflog[145] = 165; k_gflog[146] = 153; k_gflog[147] = 119; 
k_gflog[148] = 38;  k_gflog[149] = 184; k_gflog[150] = 180; k_gflog[151] = 124; 
k_gflog[152] = 17;  k_gflog[153] = 68;  k_gflog[154] = 146; k_gflog[155] = 217; 
k_gflog[156] = 35;  k_gflog[157] = 32;  k_gflog[158] = 137; k_gflog[159] = 46;

// inverse logarithm
k_gfilog[128] = 133;k_gfilog[129] = 23; k_gfilog[130] = 46; k_gfilog[131] = 92; 
k_gfilog[132] = 184;k_gfilog[133] = 109;k_gfilog[134] = 218;k_gfilog[135] = 169;
k_gfilog[136] = 79; k_gfilog[137] = 158;k_gfilog[138] = 33; k_gfilog[139] = 66;
k_gfilog[140] = 132;k_gfilog[141] = 21; k_gfilog[142] = 42; k_gfilog[143] = 84;
k_gfilog[144] = 168;k_gfilog[145] = 77; k_gfilog[146] = 154;k_gfilog[147] = 41; 
k_gfilog[148] = 82; k_gfilog[149] = 164;k_gfilog[150] = 85; k_gfilog[151] = 170;
k_gfilog[152] = 73; k_gfilog[153] = 146;k_gfilog[154] = 57; k_gfilog[155] = 114;
k_gfilog[156] = 228;k_gfilog[157] = 213;k_gfilog[158] = 183;k_gfilog[159] = 115;
}

if(tx == 5){
k_gflog[160] = 55;  k_gflog[161] = 63;  k_gflog[162] = 209; k_gflog[163] = 91;  
k_gflog[164] = 149; k_gflog[165] = 188; k_gflog[166] = 207; k_gflog[167] = 205; 
k_gflog[168] = 144; k_gflog[169] = 135; k_gflog[170] = 151; k_gflog[171] = 178; 
k_gflog[172] = 220; k_gflog[173] = 252; k_gflog[174] = 190; k_gflog[175] = 97;  
k_gflog[176] = 242; k_gflog[177] = 86;  k_gflog[178] = 211; k_gflog[179] = 171;
k_gflog[180] = 20;  k_gflog[181] = 42;  k_gflog[182] = 93;  k_gflog[183] = 158;
k_gflog[184] = 132; k_gflog[185] = 60;  k_gflog[186] = 57;  k_gflog[187] = 83;  
k_gflog[188] = 71;  k_gflog[189] = 109; k_gflog[190] = 65;  k_gflog[191] = 162; 

// inverse logarithm
k_gfilog[160] = 230;k_gfilog[161] = 209;k_gfilog[162] = 191;k_gfilog[163] = 99; 
k_gfilog[164] = 198;k_gfilog[165] = 145;k_gfilog[166] = 63; k_gfilog[167] = 126;
k_gfilog[168] = 252;k_gfilog[169] = 229;k_gfilog[170] = 215;k_gfilog[171] = 179;
k_gfilog[172] = 123;k_gfilog[173] = 246;k_gfilog[174] = 241;k_gfilog[175] = 255;
k_gfilog[176] = 227;k_gfilog[177] = 219;k_gfilog[178] = 171;k_gfilog[179] = 75;
k_gfilog[180] = 150;k_gfilog[181] = 49; k_gfilog[182] = 98; k_gfilog[183] = 196;
k_gfilog[184] = 149;k_gfilog[185] = 55; k_gfilog[186] = 110;k_gfilog[187] = 220;
k_gfilog[188] = 165;k_gfilog[189] = 87; k_gfilog[190] = 174;k_gfilog[191] = 65; 
}

if(tx == 6){
k_gflog[192] = 31;  k_gflog[193] = 45;  k_gflog[194] = 67;  k_gflog[195] = 216; 
k_gflog[196] = 183; k_gflog[197] = 123; k_gflog[198] = 164; k_gflog[199] = 118;
k_gflog[200] = 196; k_gflog[201] = 23;  k_gflog[202] = 73;  k_gflog[203] = 236;
k_gflog[204] = 127; k_gflog[205] = 12;  k_gflog[206] = 111; k_gflog[207] = 246; 
k_gflog[208] = 108; k_gflog[209] = 161; k_gflog[210] = 59;  k_gflog[211] = 82;  
k_gflog[212] = 41;  k_gflog[213] = 157; k_gflog[214] = 85;  k_gflog[215] = 170; 
k_gflog[216] = 251; k_gflog[217] = 96;  k_gflog[218] = 134; k_gflog[219] = 177;
k_gflog[220] = 187; k_gflog[221] = 204; k_gflog[222] = 62;  k_gflog[223] = 90;  

// inverse logarithm
k_gfilog[192] = 130;k_gfilog[193] = 25; k_gfilog[194] = 50; k_gfilog[195] = 100;
k_gfilog[196] = 200;k_gfilog[197] = 141;k_gfilog[198] = 7;  k_gfilog[199] = 14;
k_gfilog[200] = 28; k_gfilog[201] = 56; k_gfilog[202] = 112;k_gfilog[203] = 224;
k_gfilog[204] = 221;k_gfilog[205] = 167;k_gfilog[206] = 83; k_gfilog[207] = 166;
k_gfilog[208] = 81; k_gfilog[209] = 162;k_gfilog[210] = 89; k_gfilog[211] = 178;
k_gfilog[212] = 121;k_gfilog[213] = 242;k_gfilog[214] = 249;k_gfilog[215] = 239;
k_gfilog[216] = 195;k_gfilog[217] = 155;k_gfilog[218] = 43; k_gfilog[219] = 86;
k_gfilog[220] = 172;k_gfilog[221] = 69; k_gfilog[222] = 138;k_gfilog[223] = 9;  
}
	
if(tx == 7){
k_gflog[224] = 203; k_gflog[225] = 89;  k_gflog[226] = 95;  k_gflog[227] = 176; 
k_gflog[228] = 156; k_gflog[229] = 169; k_gflog[230] = 160; k_gflog[231] = 81;  
k_gflog[232] = 11;  k_gflog[233] = 245; k_gflog[234] = 22;  k_gflog[235] = 235; 
k_gflog[236] = 122; k_gflog[237] = 117; k_gflog[238] = 44;  k_gflog[239] = 215;
k_gflog[240] = 79;  k_gflog[241] = 174; k_gflog[242] = 213; k_gflog[243] = 233; 
k_gflog[244] = 230; k_gflog[245] = 231; k_gflog[246] = 173; k_gflog[247] = 232; 
k_gflog[248] = 116; k_gflog[249] = 214; k_gflog[250] = 244; k_gflog[251] = 234; 
k_gflog[252] = 168; k_gflog[253] = 80;  k_gflog[254] = 88;

// inverse logarithm
k_gfilog[224] = 18; k_gfilog[225] = 36; k_gfilog[226] = 72; k_gfilog[227] = 144;
k_gfilog[228] = 61; k_gfilog[229] = 122;k_gfilog[230] = 244;k_gfilog[231] = 245;
k_gfilog[232] = 247;k_gfilog[233] = 243;k_gfilog[234] = 251;k_gfilog[235] = 235;
k_gfilog[236] = 203;k_gfilog[237] = 139;k_gfilog[238] = 11; k_gfilog[239] = 22;
k_gfilog[240] = 44; k_gfilog[241] = 88; k_gfilog[242] = 176;k_gfilog[243] = 125;
k_gfilog[244] = 250;k_gfilog[245] = 233;k_gfilog[246] = 207;k_gfilog[247] = 131;
k_gfilog[248] = 27; k_gfilog[249] = 54; k_gfilog[250] = 108;k_gfilog[251] = 216;
k_gfilog[252] = 173;k_gfilog[253] = 71; k_gfilog[254] = 142;
}

__syncthreads();
	
unsigned char mult;
int i;
int j;
	
int high_disk = (disks-CHECK_SYMBOLS);
__shared__ unsigned long fetch_tmp[THREAD_BLOCKSIZE];
__shared__ unsigned long accu[THREAD_BLOCKSIZE];
		
__shared__ unsigned char *local_accu;
local_accu = (unsigned char *)&accu[tx];

__shared__ unsigned char *local_fetch_tmp;
local_fetch_tmp = (unsigned char *)&fetch_tmp[tx];

//tx = thread id im block
//bx = block id des threads
//th = thread nummer

for(j=1; j<=CHECK_SYMBOLS; j++){
	mult = j;
	accu[tx] = 0;
	
	for(i=0; i<=high_disk; i++){
		fetch_tmp[tx] = *(unsigned long *)&DEVICE_DP[(i*DMA_BLOCKSIZE)+(th*8)];
	
		local_accu[0] ^= mult_gf_shader(local_fetch_tmp[0], mult, k_gflog, k_gfilog);
		local_accu[1] ^= mult_gf_shader(local_fetch_tmp[1], mult, k_gflog, k_gfilog);
		local_accu[2] ^= mult_gf_shader(local_fetch_tmp[2], mult, k_gflog, k_gfilog);
		local_accu[3] ^= mult_gf_shader(local_fetch_tmp[3], mult, k_gflog, k_gfilog);
		local_accu[4] ^= mult_gf_shader(local_fetch_tmp[4], mult, k_gflog, k_gfilog);
		local_accu[5] ^= mult_gf_shader(local_fetch_tmp[5], mult, k_gflog, k_gfilog);
		local_accu[6] ^= mult_gf_shader(local_fetch_tmp[6], mult, k_gflog, k_gfilog);
		local_accu[7] ^= mult_gf_shader(local_fetch_tmp[7], mult, k_gflog, k_gfilog);
	
		//iterate local matrix index i^j	
		mult = mult_gf_shader(mult, j, k_gflog, k_gfilog);
		}
	*(unsigned long *)&DEVICE_PQ[((j-1)*DMA_BLOCKSIZE)+(th*8)] = accu[tx];
	}

}



/**
 * This function implements multiplication on an GF(2) with lookup tables on the
 * gpu with preinitialized lookup tables at the shared memory.
 *
 * @param a		   : first operand
 * @param b		   : second operand
 * @param gflog[]  : logarithm table
 * @param gfilog[] : inverse logarithm table
 *
 * @returns			result of the GF(2) multiplication
 */

__device__ inline unsigned char mult_gf_shader(unsigned char a, unsigned char b, unsigned char gflog[], unsigned char gfilog[])
{
unsigned char sum_log;
int w = 8;
	
if(a==0 || b==0){return 0;}
	
sum_log = gflog[a] + gflog[b];
  
if(sum_log >= NW-1){sum_log -= NW-1;}
 
return gfilog[sum_log];
}



/**
 * Get cards buffer
 *
 * @returns			void
 */

static void inline get_card_mem(void)
{

if(mem_tag == 0){
	hipMalloc((void **)&DEVICE_DP, DMA_BLOCKSIZE*256);
	hipMalloc((void **)&DEVICE_CS, DMA_BLOCKSIZE*256);
	
	hipError_t error_t;
	error_t = hipGetLastError();
	if( 0 != strcmp(hipGetErrorString(error_t), "no error") ){
		printf("Device allocation failed!\n");
		exit(1);
		}
	
	#ifdef DEBUG_LEVEL_8 
	printf("getting device memory : %s\n", cudaGetErrorString(error_t) );
	#endif
	
	mem_tag = 1;
	}

}



/**
 * Free the memory from the device
 *
 * @returns			void
 */

extern void release_card_memory(void)
{
hipFree(DEVICE_DP);
hipFree(DEVICE_CS);
}

//______________________________________________________________________________



/**
 * This function generates gflog and gfilog
 *
 * @returns			0 on success
 */

int setup_tables(void)
{
unsigned int b, log, x_to_w;

unsigned int prim_poly_8 = 0435;
unsigned int prim_poly = prim_poly_8;
int w = 8;
	
x_to_w = 1 << w;
gflog2 = (unsigned short *) malloc (sizeof(unsigned short) * x_to_w);
gfilog2 = (unsigned short *) malloc (sizeof(unsigned short) * x_to_w);

b = 1;
for (log = 0; log < x_to_w-1; log++){
	gflog2[b] = (unsigned short) log;
	gfilog2[log] = (unsigned short) b;
	b = b << 1;
	if(b & x_to_w){
	b = b ^ prim_poly;}
	}

printf("gflog[] :\n");
for (log = 0; log < x_to_w-1; log++){
	printf("%u, ",  gflog2[log]);
	}
	
printf("\ngflog[] :\n");
for (log = 0; log < x_to_w-1; log++){
	printf("k_gflog[%d] = %u;\n", log, gflog2[log]);
	}

printf("\n\ngfilog[] :\n");
for (log = 0; log < x_to_w-1; log++){
	printf("%u, ",  gfilog2[log]);
	}
	
printf("\ngflog[] :\n");
for (log = 0; log < x_to_w-1; log++){
	printf("k_gfilog[%d] = %u;\n", log, gfilog2[log]);
	}
	
return 0;
}



/**
 * The second function returns the amount of time, where the process 
 * is running. It uses the propper glibc function gettimeofday() which
 * extracts from the RTC
 *
 * @returns		Time
 */

double gtd_second(void)
{
	struct timezone tz;
	struct timeval t;
	gettimeofday(&t, &tz);

	return (double) t.tv_sec + ((double)t.tv_usec/1e6);
}
