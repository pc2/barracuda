#include "hip/hip_runtime.h"
/**
 * \file
 * \brief	Main executable for the baracuda deamon
 *
 * @author	Dominic Eschweiler weiler@upb.de
 *
 * Status	: STABLE \n
 * Date of creation : 11.5.2008
 *
 */

/*****************************************************************
 *
 * Barracuda is a experimental microdriver extension to the 
 * linux-kernel that is able to outsource common functions to
 * the userspace. It was intensionally designed to accelerate
 * CPU-intensive Tasks on a GPU.
 *
 * Copyright (C) 2009 Dominic Eschweiler
 *
 * This program is free software; you can redistribute it and/or 
 * modify it under the terms of the GNU General Public License as 
 * published by the Free Software Foundation; either only GPLv2 - 
 * version 2 of the License.
 *
 * This program is distributed in the hope that it will be useful, 
 * but WITHOUT ANY WARRANTY; without even the implied warranty of 
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the 
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public 
 * License along with this program; 
 * if not, see <http://www.gnu.org/licenses/>.
 *
 *****************************************************************/

#include <stdlib.h>
#include <stdio.h>
#include <unistd.h>
#include <string.h>
#include <dirent.h>
#include <sys/stat.h>
#include <sys/file.h>
#include <sys/types.h>
#include <time.h>
#include <sys/time.h>
#include <math.h>
#include <sched.h>
#include <signal.h>
#include <sys/wait.h>
#include <syslog.h>

#ifndef NOCUDA
	#include <hip/hip_runtime_api.h>
#endif

# include "definitions.h"
# include "service.h"
# include "vanilla/raid6vanilla.h"
# include "smp/raid6smp.h"
# include "dummy/raid6dummy.h"
# include "multrs/raid6multrs.h"
#ifndef NOCUDA
	# include "cuda/raid6cuda.h"
#endif
# include "validator.h"
# include "benchmarker.h"
# include "userspace_driver.h"

int helper();
HOST syndrome_func choose_implementation(	syndrome_func gen_syndrome,
											syndrome_func gen_syndrome_list[],
											char *mode, char **implementation_names,
											int number );


/**
 * Main Function of control
 *
 * @param argc		: # of arguments
 * @param **argv	: Array of arguments
 *
 * @returns			EXIT_FAILURE on error, EXIT_SUCCESS on no error
 */

HOST int main( int argc, char *argv[] )
{	
	/**
	 Variables
	 */
	
#ifndef NOCUDA
	/* a list of implementation-pointers */
	syndrome_func gen_syndrome_implementations[] =
	{	raid6_vanilla_gen_syndrome, 
		raid6_smp_gen_syndrome, 
		raid6_dummy_gen_syndrome,
		multi_rs_gen_syndrome,
		raid6_cuda_gen_syndrome };
	
	/* a list of corresponding implementation names */
	char *implemenatation_names[16] = 
	{ "SOFT", "SMP", "DUMMY", "MULTI", "CUDA" };

	/* How many generator functions are there */
	int number_of_generators = 5;
#endif
	
#ifdef NOCUDA
	/* a list of implementation-pointers */
	syndrome_func gen_syndrome_implementations[] =
	{	raid6_vanilla_gen_syndrome, 
		raid6_smp_gen_syndrome, 
		raid6_dummy_gen_syndrome,
		multi_rs_gen_syndrome };
	
	/* a list of corresponding implementation names */
	char *implemenatation_names[16] =
	{ "SOFT", "SMP", "DUMMY", "MULTI"};

	/* How many generator functions are there */
	int number_of_generators = 4;
#endif
	
	
	/* This is _THE_ implemenatation */
	syndrome_func gen_syndrome = gen_syndrome_implementations[0];
	
	/* The normal iterator variable */
	int i = 0;
	
	/**
	 * Process all possible arguments :
	 * -d			: deamonize
	 * -m <type>	: mode = SOFT, CUDA, FPGA
	 * -k           : kill all deamons
	 * -B <type>	: Benchmark Mode = PP_NL BW_NL PP_CB BW_CB
	 * -V			: Validation Mode ( Validate all RS implementations against the pure software Version
	 * --help -h	: show help
 	 */
	
	#ifdef DEBUG_LEVEL_1
	printf("DEBUG 1 :  There where %d(-1) arguments given\n", argc);
	#endif
	
	int  deamonize	= 0;
	int  mode		= 0;
	char mode_type[10];
	int  benchmark	= 0;
	char benchmark_type[10];
	int  validation	= 0;
	int  kill		= 0;
	int	 c_mode		= 0;
	int  rs_mode    = 0;
	
	/* Init all internal variables */
	set_internal_vars();

	/*
	 * Go through all command-line arguments and set all coresponding 
	 * configuration variables. Command.line arguments can be shown by
	 * -h or --help.
	 */
	for(i=0; i < argc; i++){
		
		#ifdef DEBUG_LEVEL_1
		printf("DEBUG 1 :  %3d : %s \n", i, argv[i]);
		#endif
		
		if( (strcmp(argv[i], "-h") == 0) || (strcmp("--help", argv[i]) == 0) ){
			helper();
			return EXIT_SUCCESS;
			}
		
		if( strcmp( argv[i], "-V") == 0 ){
			validation	= 1;
			}
		
		if( (strcmp(argv[i], "-m") == 0) && (i < argc-1) ){
			mode = 1;
			strcpy(mode_type, argv[i+1]);
			printf("Processing-mode is : %s\n", mode_type);
			if( strcmp(argv[i+1], "SOFT")  == 0 ){ rs_mode = 0; }
			if( strcmp(argv[i+1], "SMP")   == 0 ){ rs_mode = 1; }
			if( strcmp(argv[i+1], "DUMMY") == 0 ){ rs_mode = 2; }
			if( strcmp(argv[i+1], "MULTI")  == 0 ){ rs_mode = 3; }
			if( strcmp(argv[i+1], "CUDA")  == 0 ){ rs_mode = 4; }
			}
		
		if( (strcmp(argv[i], "-B") == 0) && (i < argc-1) ){
			benchmark = 1;
			strcpy(benchmark_type, argv[i+1]);
			printf("Benchmark-mode is  : %s\n", benchmark_type);
			}
		
		if( strcmp(argv[i], "-d") == 0 ){
			deamonize = 1;
			printf("Deamon mode activated.\n");
			}
		
		if( strcmp(argv[i], "-k") == 0 ){
			kill = 1;
			printf("Deamon would be terminated.\n");
			}
		
		if( strcmp(argv[i], "-c") == 0 ){
			if( strcmp(argv[i+1], "NL")    == 0 ){ c_mode = 1; }
			if( strcmp(argv[i+1], "IOCTL") == 0 ){ c_mode = 2; }
			if( strcmp(argv[i+1], "PFS")   == 0 ){ c_mode = 3; }
			}
		}

	/*
	 * In case that the user calles stop deamon, this handler lead to a managed 
	 * state.
	 */
	FILE *fp;
	pid_t pid;
	char kill_command[100];
	if( kill == 1){
		fp = fopen("/tmp/baracuda_pid", "r");
		if(fp == NULL){
			printf("Can't open Pidfile\n");
			return EXIT_FAILURE;
			}
		
		fread( (void *)&pid, sizeof(pid_t), 1, fp );
		fclose(fp);	
		
		printf("PID is : %d \n", pid);
		
		/* kill, however, does not work!?*/
		//kill(pid, SIGALRM);
		
		/* this is a quick fix which works properly */
		sprintf( (char *)&kill_command, "%s%d", "/bin/kill -SIGALRM ", pid );
		printf("Kill command is : %s\n", kill_command);
		system(kill_command);
		
		return EXIT_SUCCESS;
		}

	/*
	 * A valid mode must be allways choosen, therefore search for a corresponding
	 * implementation to the input string.
	 */
	
	if(mode == 0){
		printf("No valid mode was set. Please set a mode with -m\n");
		printf("See -h for valid modes ...\n");
		return EXIT_FAILURE;
		}
	else{
		gen_syndrome = choose_implementation( gen_syndrome, gen_syndrome_implementations, 
							   mode_type, implemenatation_names, number_of_generators );
			
		}
	
	/*
	 * If the validation flag was choosen, run the validator. This validator checks
	 * if the given RS implementation does the same as the default native software
	 * implementation. This pease of code is located in validator.c ( and .h)
	 */
	
	if( validation == 1){

		if( validate_implemenataion( gen_syndrome, gen_syndrome_implementations) == EXIT_SUCCESS ){ 
			printf("Output is correct\n");
			return EXIT_SUCCESS;
			}
		else{
			printf("!!! The output from the choosen implemenation is not valid !!!\n");
			return EXIT_FAILURE;
			}

		}

	/*
	 * If the benchmark flag was choosen, run the benchmarker subroutine. The
	 * variable <benchmark_type> sets the related benchmark. A list of available
	 * benchmarks which are included in the deamon could be get by the command-
	 * line-argument --help or -h
	 */
	
	if( benchmark == 1){
		baracuda_benchmarker( 	benchmark_type, gen_syndrome_implementations, 
								implemenatation_names, number_of_generators, rs_mode);
		}
	
	/*
	 * If the deamonize flag was choosen, [clone] and therefore deamonize. The 
	 * related subroutines are implemented in userspace_driver.cu (and .h). This
	 * last function call starts the actual device driver for the RS-Calculations.
	 */

	thread_container tc;
	tc.c_mode = c_mode;
	tc.gen_syndrome = gen_syndrome;
	
	if( deamonize == 1){
		if(c_mode == 0){
			printf("No valid connection-mode was chosen, see -h or --help for all possible connection-types\n");
			}
		
		if( clone(&userspace_driver_main, &(stack[10000]), CLONE_VM | SIGCHLD, (void *)&tc) == -1 ){
			printf("Barracuda daemoninzing failed -> cloning failed\n");
			return EXIT_FAILURE;
			}
		printf("Daemon-Mode was called. Please check your logfile (maybe /var/log/messages) for success!\n");
		}
	else{
		printf("Foreground-Mode was called.\n");
		userspace_driver_main((void *)&tc);
		}
	
	return EXIT_SUCCESS;
}



/**
 * Choose the implementation corresponding to the mode
 *
 * @param gen_syndrome					: Target function pointer
 * @param gen_syndrome_list[]			: Array pointer to the availaible implementations
 * @param *mode							: Mode string
 * @param **implementation_names		: Names coresponding to the function pointers
 * @param number						: Number of implemenations
 *
 * @returns			Function pointer to the choosen implementation
 */

HOST syndrome_func choose_implementation(	syndrome_func gen_syndrome,
											syndrome_func gen_syndrome_list[],
											char *mode, char **implementation_names,
											int number )
{
	int i;

	gen_syndrome = gen_syndrome_list[0];

	for(i = 0; i < number; i++){
		if(strcmp( mode, implementation_names[i]) == 0){
			gen_syndrome = gen_syndrome_list[i];
			printf("%s as implementation was choosen!\n", implementation_names[i]);
			return gen_syndrome;
			}
		}
	
printf("No valid mode was set. Please set a mode with -m\n");
printf("See -h for valid modes ...\n Fallback to Software!\n");
return gen_syndrome;
}



/**
 * This functions prints only a help statement to the shell
 *
 * @returns			EXIT_FAILURE on error, EXIT_SUCCESS on no error
 */

HOST int helper()
{
	printf("This is the baracuda deamon.\n");
	printf("This machine is a %d bit architecture!\n", BITS_PER_LONG );
	printf(" --help -h    : show this help\n");
	printf(" -d           : deamonize\n");
	printf(" -k           : kill all deamons\n");
	printf(" -m <mode>    : Reed-Solomon implementation mode\n");
#ifndef NOCUDA
	printf("Valid modes are SOFT, CUDA, MULTI, SMP\n");
#endif
#ifdef NOCUDA
	printf("Valid modes are SOFT, MULTI, SMP\n");
#endif	
	printf(" -c <mode>    : Setup the connection mode\n");
	printf("Valid modes are NL, IOCTL, PFS\n");
	printf(" -V           : Validation-mode (Validate the choosen RS implementations against the pure software-version)\n");
	printf(" -B <mode>    : Benchmark-mode\n");
	printf("Valid modes are	DRYRUN, CUDA_BANDWIDTH, CUDA_XOR, CUDA_SHIFT\n");
	
	return EXIT_SUCCESS;
}


