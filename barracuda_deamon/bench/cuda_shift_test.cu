#include "hip/hip_runtime.h"
/**
 * \file
 * \brief	Benchmarking function for the CUDA bitshifting implementation
 *
 * @author	Dominic Eschweiler weiler@upb.de
 *
 * Status	: STABLE\n
 * Date of creation : 7.8.2008
 *
 */

/*****************************************************************
 *
 * Barracuda is a experimental microdriver extension to the 
 * linux-kernel that is able to outsource common functions to
 * the userspace. It was intensionally designed to accelerate
 * CPU-intensive Tasks on a GPU.
 *
 * Copyright (C) 2009 Dominic Eschweiler
 *
 * This program is free software; you can redistribute it and/or 
 * modify it under the terms of the GNU General Public License as 
 * published by the Free Software Foundation; either only GPLv2 - 
 * version 2 of the License.
 *
 * This program is distributed in the hope that it will be useful, 
 * but WITHOUT ANY WARRANTY; without even the implied warranty of 
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the 
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public 
 * License along with this program; 
 * if not, see <http://www.gnu.org/licenses/>.
 *
 *****************************************************************/

# include "cuda_shift_test.h"

#include <stdlib.h>
#include <stdio.h>
#include <unistd.h>
#include <string.h>
#include <dirent.h>
#include <math.h>
#include <time.h>

#include <sys/time.h>
#include <sys/mman.h>
#include <sys/stat.h>
#include <sys/file.h>
#include <sys/types.h>

#include "../service.h"

/* DEFINES */
# define BL_SIZE  10000000


#define THREAD_BLOCKSIZE 256

__host__ void tester_shift( unsigned long bl_size);
__host__ void shift_it_cpu( char *A, char *C, unsigned long bl_size );
__global__ void shift_it_cuda( char *A, char *C );



#ifndef MAIN_IS_ACTIVE
int main()
{
	test_cuda_shift_perf();
	return 0;
}
#endif



/**
 * Main routine which tests the SHIFT function for all defined block sizes. The
 * intervall is defined by the two preprocessors BL_SIZE_START and BL_SIZE_STOP
 *
 * @returns			void
 */

__host__ void test_cuda_shift_perf()
{
int i;
for(i=0; i < 30; i++){
	tester_shift(BL_SIZE);
	}
}



/**
 * This function tests bitshifting for CUDA with synchronious copys,
 * CUDA with asynchronious copys and on the CPU for a defined DMA
 * blocksize and prints the results as CSV (comma separated values)
 * on the screen.
 *
 * @param bl_size	: Blocksize of each data transfer
 *
 * @returns		void
 */

__host__ void tester_shift( unsigned long bl_size)
{
	double time_all = 0;
	double time_one = 0;
	unsigned long runs = 0;

	/* get Memory on the device */
	char *DEVICE_A;
	char *DEVICE_C;
	hipMalloc((void**)&DEVICE_A, bl_size);
	hipMalloc((void**)&DEVICE_C, bl_size);
	
	char *A = (char *)malloc(bl_size);
	char *C = (char *)malloc(bl_size);
	
	/* define thread grid dimension */
	dim3 dimBlock;
	dim3 dimGrid;
	
	dimBlock.x=THREAD_BLOCKSIZE;
	dimBlock.y=1;
	dimBlock.z=1;
	dimGrid.x=(bl_size/(dimBlock.x*8));
	dimGrid.y=1;
	
	/*----reset-vars---------------------------------*/
	runs            = 0;
	time_one		= 0;
	time_all        = 0;
	
	/*----test-gpu-performance-----------------------*/
	while(time_all < 1){
		runs++;
		time_one = gtd_second();

		shift_it_cuda<<<dimGrid, dimBlock>>>(DEVICE_A, DEVICE_C);

		time_all = time_all + (gtd_second() - time_one);
		}
	printf("GPU ; %f\n", (double)((runs*bl_size)/time_all) );
	
	/*
	hipError_t error_t;
	error_t = hipGetLastError();
	printf("kernel : %s\n", hipGetErrorString(error_t) );
	*/
	
	/*----reset-vars---------------------------------*/
	runs            = 0;
	time_one		= 0;
	time_all        = 0;

	/*----test-cpu-performance----------------------*/	
	while(time_all < 1){
		runs++;
		time_one = gtd_second();
			shift_it_cpu(A, C, bl_size);
		time_all = time_all + (gtd_second() - time_one);
		}
	printf("CPU ; %f\n", (double)((runs*bl_size)/time_all) );

	/*------------------------------------------------------------------------*/

	/* Free memory */
	hipFree(DEVICE_A);
	hipFree(DEVICE_C);
	free(A);
	free(C);
}



/**
 * Bitshift compute kernel for the main CPU
 *
 * @param *A			: A is an array of bytes for the input
 * @param *C			: C is an array of bytes for the output
 * @param bl_size		: # of bytes that should be shifted
 *
 * @returns		void
 */

__host__ void shift_it_cpu( char *A, char *C, unsigned long bl_size )
{
unsigned long i = 0;

unsigned long *TMP_A = (unsigned long *)A;
unsigned long *TMP_C = (unsigned long *)C;
	
for(i=0; i < (bl_size/sizeof(unsigned long)); i++){
	TMP_C[i] = TMP_A[i] << 1;
	}
}



/**
 * Bitshift compute kernel for the main GPU
 *
 * @param *A			: A is an array of bytes for the input
 * @param *C			: C is an array of bytes for the output
 *
 * @returns		void
 */

__global__ void shift_it_cuda( char *A, char *C )
{
	int bx = blockIdx.x;
	int tx = threadIdx.x;
	int d  = (bx*THREAD_BLOCKSIZE)+tx;
	
	unsigned long *tmp_a, *tmp_c;
	
	tmp_a = (unsigned long *)A;
	tmp_c = (unsigned long *)C;
	
	__shared__ unsigned long a_stream[8];
	__shared__ unsigned long c_stream[8];
	
	c_stream[tx] = tmp_c[d];
	a_stream[tx] = c_stream[tx] << 1;
	tmp_a[d]     = a_stream[tx];	
}
