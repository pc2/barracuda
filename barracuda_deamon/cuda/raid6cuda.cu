#include "hip/hip_runtime.h"
/**
 * \file
 * \brief	Cuda implementation of the raid6 userspace functions
 *
 * @author	Dominic Eschweiler weiler@upb.de
 *
 * Status	: STABLE\n
 * Date of creation : 19.5.2008
 *
 */

/*****************************************************************
 *
 * Barracuda is a experimental microdriver extension to the 
 * linux-kernel that is able to outsource common functions to
 * the userspace. It was intensionally designed to accelerate
 * CPU-intensive Tasks on a GPU.
 *
 * Copyright (C) 2009 Dominic Eschweiler
 *
 * This program is free software; you can redistribute it and/or 
 * modify it under the terms of the GNU General Public License as 
 * published by the Free Software Foundation; either only GPLv2 - 
 * version 2 of the License.
 *
 * This program is distributed in the hope that it will be useful, 
 * but WITHOUT ANY WARRANTY; without even the implied warranty of 
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the 
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public 
 * License along with this program; 
 * if not, see <http://www.gnu.org/licenses/>.
 *
 *****************************************************************/

#include <stdlib.h>
#include <stdio.h>
#include <unistd.h>
#include <string.h>
#include <dirent.h>
#include <sys/stat.h>
#include <sys/file.h>
#include <sys/types.h>
#include <time.h>
#include <sys/time.h>
#include <math.h>
#include <syslog.h>

#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

#include "raid6cuda.h"

#define INHOST_COPY
#define CUDA_COPY
//#define ASYNC
#define KERNEL_EXEC

#define DMA_BLOCKSIZE 524288
#define THREAD_BLOCKSIZE 256

#define NBYTES_CUDA(x) ((x) * 0x0101010101010101UL)

static u8 *DEVICE_DP_1;
static u8 *DEVICE_PQ_1;
	
static u8 *DEVICE_DP_2;
static u8 *DEVICE_PQ_2;

#ifdef ASYNC
	static u8 *HOST_DP_1;
	static u8 *HOST_PQ_1;
	
	static u8 *HOST_DP_2;
	static u8 *HOST_PQ_2;
#endif

static int mem_tag = 0;

// function prototypes
#ifdef ASYNC
static void raid6_cuda_gen_syndrome_asynccopy(int disks, size_t bytes, void **ptrs);
#endif
#ifndef ASYNC
static void raid6_cuda_gen_syndrome_synccopy(int disks, size_t bytes, void **ptrs);
#endif

static void inline get_card_mem(void);

__global__ void syndrome_block( u8 *DEVICE_DP, u8 *DEVICE_PQ, int z0);


/**
 * This is NVIDIA CUDA version of gen_syndrome
 *
 * @param disks		: # of disks
 * @param bytes		: # number of bytes
 * @param **ptrs	: processing data
 *
 * @returns			void
 */

extern void raid6_cuda_gen_syndrome(int disks, size_t bytes, void **ptrs)
{	 
#ifdef ASYNC
	raid6_cuda_gen_syndrome_asynccopy(disks, bytes, ptrs);
#endif
	
#ifndef ASYNC
	raid6_cuda_gen_syndrome_synccopy(disks, bytes, ptrs);
#endif
}



/**
 * This is NVIDIA CUDA version of gen_syndrome, which uses asynchronious copy
 * and execution.
 *
 * @param disks		: # of disks
 * @param bytes		: # number of bytes
 * @param **ptrs	: processing data
 *
 * @returns			void
 */
#ifdef ASYNC
static void raid6_cuda_gen_syndrome_asynccopy(int disks, size_t bytes, void **ptrs)
{
dim3 dimBlock;
dim3 dimGrid;
	
// variables
int i, j;
static u8 **dptrs = (u8 **)ptrs;

get_card_mem();
	
unsigned long runs = floor(bytes/(DMA_BLOCKSIZE*2));
unsigned long carry_off = 0;

dimBlock.x=THREAD_BLOCKSIZE;
dimBlock.y=1;
dimBlock.z=1;
dimGrid.x=(DMA_BLOCKSIZE/(dimBlock.x*8));
dimGrid.y=1;
	
hipStream_t stream[2];
	
#ifdef DEBUG_LEVEL_8
	printf("block_x : %d, dimgrid_x : %d, runs : %d\n", dimBlock.x, dimGrid.x, runs);
	hipError_t error_t;
	error_t = hipGetLastError();
	printf("stream create : %s\n", hipGetErrorString(error_t) );
#endif

u8 *tmp;
	
for(j=0; j<runs; j++){
	hipStreamCreate(&stream[0]);
	hipStreamCreate(&stream[1]);
	
	// Copy the stuff to the page locked buffer
	for(i=0; i<disks-2; i++){
		#ifdef INHOST_COPY
		tmp = dptrs[i];
		hipMemcpy(	&HOST_DP_1[i*DMA_BLOCKSIZE], 
					&tmp[(j*2*DMA_BLOCKSIZE)], 
					DMA_BLOCKSIZE, 
					hipMemcpyHostToHost);

		hipMemcpy(	&HOST_DP_2[i*DMA_BLOCKSIZE], 
					&tmp[(j*2*DMA_BLOCKSIZE)+DMA_BLOCKSIZE], 
					DMA_BLOCKSIZE, 
					hipMemcpyHostToHost);
		#endif
		}
	#ifdef DEBUG_LEVEL_8
	error_t = hipGetLastError();
	printf("inhost copy to : %s\n", hipGetErrorString(error_t) );
	#endif
	
	#ifdef CUDA_COPY
	hipMemcpyAsync( DEVICE_DP_1, HOST_DP_1, DMA_BLOCKSIZE*(disks-2), hipMemcpyHostToDevice, stream[0]);
	hipMemcpyAsync( DEVICE_DP_2, HOST_DP_2, DMA_BLOCKSIZE*(disks-2), hipMemcpyHostToDevice, stream[1]);
	#endif
	
	#ifdef DEBUG_LEVEL_8
	error_t = hipGetLastError();
	printf("copy to device : %s\n", hipGetErrorString(error_t) );
	#endif
	
	#ifdef KERNEL_EXEC
	syndrome_block<<<dimGrid, dimBlock, stream[0]>>>( DEVICE_DP_1, DEVICE_PQ_1, disks-3);
	syndrome_block<<<dimGrid, dimBlock, stream[1]>>>( DEVICE_DP_2, DEVICE_PQ_2, disks-3);
	#endif
	
	#ifndef KERNEL_EXEC
	hipDeviceSynchronize();
	#endif
	
	#ifdef DEBUG_LEVEL_8 
		error_t = hipGetLastError();
		printf("kernel exec : %s\n", hipGetErrorString(error_t) );
	#endif
	
	#ifdef CUDA_COPY
	hipMemcpyAsync( HOST_PQ_1, DEVICE_PQ_1, DMA_BLOCKSIZE*2, hipMemcpyDeviceToHost, stream[0]);
	hipMemcpyAsync( HOST_PQ_2, DEVICE_PQ_2, DMA_BLOCKSIZE*2, hipMemcpyDeviceToHost, stream[1]);
	#endif
	
	#ifdef DEBUG_LEVEL_8
	error_t = hipGetLastError();
	printf("copy from device : %s\n", hipGetErrorString(error_t) );
	#endif
	
	hipDeviceSynchronize();
	
	#ifdef INHOST_COPY
	hipMemcpy(&dptrs[disks-1][j*2*DMA_BLOCKSIZE], HOST_PQ_1, DMA_BLOCKSIZE, hipMemcpyHostToHost);
	hipMemcpy(&dptrs[disks-2][j*2*DMA_BLOCKSIZE], &HOST_PQ_1[1], DMA_BLOCKSIZE, hipMemcpyHostToHost);
	
	hipMemcpy(&dptrs[disks-1][(j*2*DMA_BLOCKSIZE)+DMA_BLOCKSIZE], HOST_PQ_2, DMA_BLOCKSIZE, hipMemcpyHostToHost);
	hipMemcpy(&dptrs[disks-2][(j*2*DMA_BLOCKSIZE)+DMA_BLOCKSIZE], &HOST_PQ_2[1], DMA_BLOCKSIZE, hipMemcpyHostToHost);
	#endif
	
	#ifdef DEBUG_LEVEL_8
	error_t = hipGetLastError();
	printf("copy PQ inhost : %s\n", hipGetErrorString(error_t) );
	#endif
		
	hipStreamDestroy(stream[0]);
	hipStreamDestroy(stream[1]);
	}
	
// This is for the part which fits not into a 2*DMA_BLOCKSIZE big block
// but in a DMA_BLOCKSIZE big block
carry_off = bytes-(runs * 2 * DMA_BLOCKSIZE);
if(carry_off >= DMA_BLOCKSIZE){
	hipStreamCreate(&stream[0]);
	
	// Copy the stuff to the page locked buffer
	for(i=0; i<disks-2; i++){
		#ifdef INHOST_COPY
		tmp = dptrs[i];
		hipMemcpy(	&HOST_DP_1[i*DMA_BLOCKSIZE], 
					&tmp[(j*2*DMA_BLOCKSIZE)], 
					DMA_BLOCKSIZE, 
					hipMemcpyHostToHost);
		#endif
		}
	#ifdef DEBUG_LEVEL_8
	error_t = hipGetLastError();
	printf("inhost copy to : %s\n", hipGetErrorString(error_t) );
	#endif
	
	#ifdef CUDA_COPY
	hipMemcpyAsync( DEVICE_DP_1, HOST_DP_1, DMA_BLOCKSIZE*(disks-2), hipMemcpyHostToDevice, stream[0]);
	#endif
	
	#ifdef DEBUG_LEVEL_8
	error_t = hipGetLastError();
	printf("copy to device : %s\n", hipGetErrorString(error_t) );
	#endif
	
	#ifdef KERNEL_EXEC
	syndrome_block<<<dimGrid, dimBlock, stream[0]>>>( DEVICE_DP_1, DEVICE_PQ_1, disks-3);
	#endif
	
	#ifndef KERNEL_EXEC
	hipDeviceSynchronize();
	#endif
	
	#ifdef DEBUG_LEVEL_8 
		error_t = hipGetLastError();
		printf("kernel exec : %s\n", hipGetErrorString(error_t) );
	#endif
	
	#ifdef CUDA_COPY
	hipMemcpyAsync( HOST_PQ_1, DEVICE_PQ_1, DMA_BLOCKSIZE*2, hipMemcpyDeviceToHost, stream[0]);
	#endif
	
	#ifdef DEBUG_LEVEL_8
	error_t = hipGetLastError();
	printf("copy from device : %s\n", hipGetErrorString(error_t) );
	#endif
	
	hipDeviceSynchronize();
	
	#ifdef INHOST_COPY
	hipMemcpy(&dptrs[disks-1][j*2*DMA_BLOCKSIZE], HOST_PQ_1, DMA_BLOCKSIZE, hipMemcpyHostToHost);
	hipMemcpy(&dptrs[disks-1][(j*2*DMA_BLOCKSIZE)+DMA_BLOCKSIZE], HOST_PQ_2, DMA_BLOCKSIZE, hipMemcpyHostToHost);
	#endif
	
	#ifdef DEBUG_LEVEL_8
	error_t = hipGetLastError();
	printf("copy PQ inhost : %s\n", hipGetErrorString(error_t) );
	#endif
		
	hipStreamDestroy(stream[0]);

	j=(j*2)+1;
	carry_off = carry_off-DMA_BLOCKSIZE;
	}
	
	
// This is for the last part which fits not into a DMA_BLOCKSIZE big block.
if(carry_off > 0){
	//-------------------------------------------------------------------
	hipStreamCreate(&stream[0]);
	
	// Copy the stuff to the page locked buffer
	for(i=0; i<disks-2; i++){
		#ifdef INHOST_COPY
		tmp = dptrs[i];
		hipMemcpy(	&HOST_DP_1[i*DMA_BLOCKSIZE], 
					&tmp[j*DMA_BLOCKSIZE], 
					DMA_BLOCKSIZE, 
					hipMemcpyHostToHost);
		#endif
		}
	#ifdef DEBUG_LEVEL_8
	error_t = hipGetLastError();
	printf("inhost copy to : %s\n", hipGetErrorString(error_t) );
	#endif
	
	#ifdef CUDA_COPY
	hipMemcpyAsync( DEVICE_DP_1, HOST_DP_1, DMA_BLOCKSIZE*(disks-2), hipMemcpyHostToDevice, stream[0]);
	#endif
	
	#ifdef DEBUG_LEVEL_8
	error_t = hipGetLastError();
	printf("copy to device : %s\n", hipGetErrorString(error_t) );
	#endif
	
	#ifdef KERNEL_EXEC
	dimBlock.x=THREAD_BLOCKSIZE;
	dimBlock.y=1;
	dimBlock.z=1;
	dimGrid.x=(carry_off/(dimBlock.x*8))+1;
	dimGrid.y=1;
	syndrome_block<<<dimGrid, dimBlock, stream[0]>>>( DEVICE_DP_1, DEVICE_PQ_1, disks-3);
	#endif

	#ifndef KERNEL_EXEC
	hipDeviceSynchronize();
	#endif
	
	#ifdef DEBUG_LEVEL_8 
		error_t = hipGetLastError();
		printf("kernel exec : %s\n", hipGetErrorString(error_t) );
	#endif
	
	#ifdef CUDA_COPY
	hipMemcpyAsync( HOST_PQ_1, DEVICE_PQ_1, DMA_BLOCKSIZE*2, hipMemcpyDeviceToHost, stream[0]);
	#endif
	
	#ifdef DEBUG_LEVEL_8
	error_t = hipGetLastError();
	printf("copy from device : %s\n", hipGetErrorString(error_t) );
	#endif
	
	hipDeviceSynchronize();
	
	#ifdef INHOST_COPY
	hipMemcpy(&dptrs[disks-1][j*2*DMA_BLOCKSIZE], HOST_PQ_1, DMA_BLOCKSIZE, hipMemcpyHostToHost);
	hipMemcpy(&dptrs[disks-1][(j*2*DMA_BLOCKSIZE)+DMA_BLOCKSIZE], HOST_PQ_2, DMA_BLOCKSIZE, hipMemcpyHostToHost);
	#endif
	
	#ifdef DEBUG_LEVEL_8
	error_t = hipGetLastError();
	printf("copy PQ inhost : %s\n", hipGetErrorString(error_t) );
	#endif
		
	hipStreamDestroy(stream[0]);
	
	}

}
#endif


/**
 * This is NVIDIA CUDA version of gen_syndrome, which uses synchronious copy and 
 * execution.
 *
 * @param disks		: # of disks
 * @param bytes		: # number of bytes
 * @param **ptrs	: processing data
 *
 * @returns			void
 */
#ifndef ASYNC
static void raid6_cuda_gen_syndrome_synccopy(int disks, size_t bytes, void **ptrs)
{
dim3 dimBlock;
dim3 dimGrid;
	
/* variables */
int i, j;
static u8 **dptrs = (u8 **)ptrs;

get_card_mem();
	
unsigned long runs = floor(bytes/(DMA_BLOCKSIZE*2));
unsigned long carry_off = 0;

dimBlock.x=THREAD_BLOCKSIZE;
dimBlock.y=1;
dimBlock.z=1;
dimGrid.x=(DMA_BLOCKSIZE/(dimBlock.x*8));
dimGrid.y=1;
	
#ifdef DEBUG_LEVEL_8
	printf("block_x : %d, dimgrid_x : %d, runs : %d\n", dimBlock.x, dimGrid.x, runs);
	hipError_t error_t;
#endif

for(j=0; j<runs; j++){	
	#ifdef CUDA_COPY
	for(i=0; i<disks-2; i++){		
		hipMemcpy( &DEVICE_DP_1[i*DMA_BLOCKSIZE], 
				   	&dptrs[i][j*2*DMA_BLOCKSIZE],
					DMA_BLOCKSIZE, 
					hipMemcpyHostToDevice);
		
		hipMemcpy( &DEVICE_DP_2[i*DMA_BLOCKSIZE], 
				   	&dptrs[i][(j*2*DMA_BLOCKSIZE)+DMA_BLOCKSIZE],
					DMA_BLOCKSIZE, 
					hipMemcpyHostToDevice);
		}
	#endif
	
	#ifdef DEBUG_LEVEL_8
	error_t = hipGetLastError();
	printf("todev copy to : %s\n", hipGetErrorString(error_t) );
	#endif

	#ifdef KERNEL_EXEC
	syndrome_block<<<dimGrid, dimBlock>>>( DEVICE_DP_1, DEVICE_PQ_1, disks-3);
	syndrome_block<<<dimGrid, dimBlock>>>( DEVICE_DP_2, DEVICE_PQ_2, disks-3);
	#endif
	
	hipDeviceSynchronize();
	
	#ifdef DEBUG_LEVEL_8 
		error_t = hipGetLastError();
		printf("kernel exec : %s\n", hipGetErrorString(error_t) );
	#endif
		
	#ifdef CUDA_COPY
	hipMemcpy(&dptrs[disks-2][j*2*DMA_BLOCKSIZE], 
			   DEVICE_PQ_1, DMA_BLOCKSIZE, 
			   hipMemcpyDeviceToHost);
	hipMemcpy(&dptrs[disks-1][j*2*DMA_BLOCKSIZE], 
			   &DEVICE_PQ_1[DMA_BLOCKSIZE], DMA_BLOCKSIZE, 
			   hipMemcpyDeviceToHost);
	
	hipMemcpy(&dptrs[disks-2][(j*2*DMA_BLOCKSIZE)+DMA_BLOCKSIZE], 
			   DEVICE_PQ_2, DMA_BLOCKSIZE, hipMemcpyDeviceToHost);
	hipMemcpy(&dptrs[disks-1][(j*2*DMA_BLOCKSIZE)+DMA_BLOCKSIZE], 
			   &DEVICE_PQ_2[DMA_BLOCKSIZE], DMA_BLOCKSIZE, hipMemcpyDeviceToHost);
	#endif
	
	#ifdef DEBUG_LEVEL_8
	error_t = hipGetLastError();
	printf("fromdev copy : %s\n", hipGetErrorString(error_t) );
	#endif
	}
	
// This is for the part which fits not into a 2*DMA_BLOCKSIZE big block
// but in a DMA_BLOCKSIZE big block
carry_off = bytes-(runs * 2 * DMA_BLOCKSIZE);
if(carry_off >= DMA_BLOCKSIZE){
	#ifdef CUDA_COPY
	for(i=0; i<disks-2; i++){		
		hipMemcpy( &DEVICE_DP_1[i*DMA_BLOCKSIZE], 
				   	&dptrs[i][j*2*DMA_BLOCKSIZE],
					DMA_BLOCKSIZE, 
					hipMemcpyHostToDevice);
		}
	#endif
	
	#ifdef DEBUG_LEVEL_8
	error_t = hipGetLastError();
	printf("todev copy to : %s\n", hipGetErrorString(error_t) );
	#endif

	#ifdef KERNEL_EXEC
	syndrome_block<<<dimGrid, dimBlock>>>( DEVICE_DP_1, DEVICE_PQ_1, disks-3);
	#endif
	
	hipDeviceSynchronize();
	
	#ifdef DEBUG_LEVEL_8 
		error_t = hipGetLastError();
		printf("kernel exec : %s\n", hipGetErrorString(error_t) );
	#endif
		
	#ifdef CUDA_COPY
	hipMemcpy(&dptrs[disks-2][j*2*DMA_BLOCKSIZE], 
			   DEVICE_PQ_1, DMA_BLOCKSIZE, 
			   hipMemcpyDeviceToHost);
	hipMemcpy(&dptrs[disks-1][j*2*DMA_BLOCKSIZE], 
			   &DEVICE_PQ_1[DMA_BLOCKSIZE], DMA_BLOCKSIZE, 
			   hipMemcpyDeviceToHost);
	#endif
	
	#ifdef DEBUG_LEVEL_8
	error_t = hipGetLastError();
	printf("fromdev copy : %s\n", hipGetErrorString(error_t) );
	#endif
		
	j=(j*2)+1;
	carry_off = carry_off-DMA_BLOCKSIZE;
	}

// This is for the last part which fits not into a DMA_BLOCKSIZE big block.
if(carry_off > 0){
	#ifdef CUDA_COPY
	for(i=0; i<disks-2; i++){		
		hipMemcpy( &DEVICE_DP_1[i*DMA_BLOCKSIZE], 
				   	&dptrs[i][j*DMA_BLOCKSIZE],
					carry_off, 
					hipMemcpyHostToDevice);
		}
	#endif
	
	#ifdef DEBUG_LEVEL_8
	error_t = hipGetLastError();
	printf("todev copy to : %s\n", hipGetErrorString(error_t) );
	#endif

	#ifdef KERNEL_EXEC
	dimBlock.x=THREAD_BLOCKSIZE;
	dimBlock.y=1;
	dimBlock.z=1;
	dimGrid.x=(carry_off/(dimBlock.x*8))+1;
	dimGrid.y=1;	
	syndrome_block<<<dimGrid, dimBlock>>>( DEVICE_DP_1, DEVICE_PQ_1, disks-3);
	#endif
	
	hipDeviceSynchronize();
	
	#ifdef DEBUG_LEVEL_8 
		error_t = hipGetLastError();
		printf("kernel exec : %s\n", hipGetErrorString(error_t) );
	#endif
		
	#ifdef CUDA_COPY
	hipMemcpy(&dptrs[disks-2][j*DMA_BLOCKSIZE], 
			   DEVICE_PQ_1, carry_off, 
			   hipMemcpyDeviceToHost);
	hipMemcpy(&dptrs[disks-1][j*DMA_BLOCKSIZE], 
			   &DEVICE_PQ_1[DMA_BLOCKSIZE], carry_off, 
			   hipMemcpyDeviceToHost);
	#endif
	
	#ifdef DEBUG_LEVEL_8
	error_t = hipGetLastError();
	printf("fromdev copy : %s\n", hipGetErrorString(error_t) );
	#endif
	}
	
}
#endif



/**
 * In CUDA each iteration of a loop can be expressed as a thread. This function
 * generate the syndrome for one byte vector of each disk. 
 *
 * @param *DEVICE_DP : marshalled data from the discs, which are represented in a linearized array
 * @param *DEVICE_PQ : XOR Parity and RS double parity
 * @param z0         : # of the highest data-disk
 *
 * @returns			void
 */

__global__ void syndrome_block( u8 *DEVICE_DP, u8 *DEVICE_PQ, int z0)
{
/*
int gdx = gridDim.x;
int gdy = gridDim.y;

int bdx = blockDim.x;
int bdy = blockDim.y;
int bdz = blockDim.z;
	
int by = blockIdx.y;
int bz = blockIdx.z;
	
int ty = threadIdx.y;
*/

int bx = blockIdx.x;
int tx = threadIdx.x;
int d  = ((bx*THREAD_BLOCKSIZE)+tx)*8;
	
int z;

__shared__ unsigned long wd0[THREAD_BLOCKSIZE];
__shared__ unsigned long wq0[THREAD_BLOCKSIZE]; 
__shared__ unsigned long wp0[THREAD_BLOCKSIZE]; 
__shared__ unsigned long w10[THREAD_BLOCKSIZE];
__shared__ unsigned long w20[THREAD_BLOCKSIZE];

u8 *p, *q;
p = DEVICE_PQ;
q = &DEVICE_PQ[DMA_BLOCKSIZE];

//for ( d = 0; d < bytes; d += NSIZE ){
wq0[tx] = wp0[tx] = *(unsigned long *)&DEVICE_DP[(z0*DMA_BLOCKSIZE)+d];
for ( z = z0-1; z >= 0; z-- ){
	wd0[tx] = *(unsigned long *)&DEVICE_DP[(z*DMA_BLOCKSIZE)+d];
	wp0[tx] ^= wd0[tx];
	
	//w20 = MASK(wq0);
	wq0[tx] = wq0[tx] & NBYTES_CUDA(0x80);
	wq0[tx] = (wq0[tx] << 1) - (wq0[tx] >> 7);
		
	//w10 = SHLBYTE(wq0);
	w10[tx] = (wq0[tx] << 1) & NBYTES_CUDA(0xfe);
	
	w20[tx] &= NBYTES_CUDA(0x1d);
	w10[tx] ^= w20[tx];
	wq0[tx] = w10[tx] ^ wd0[tx];
	}
	
*(unsigned long *)&p[d] = wp0[tx];
*(unsigned long *)&q[d] = wq0[tx];

//*(unsigned long *)&p[d] = 4;
//*(unsigned long *)&q[d] = 8;

}



static void inline get_card_mem(void)
{
if(mem_tag == 0){
	hipMalloc((void **)&DEVICE_DP_1, DMA_BLOCKSIZE*256);
	hipMalloc((void **)&DEVICE_PQ_1, DMA_BLOCKSIZE*2);
	
	hipMalloc((void **)&DEVICE_DP_2, DMA_BLOCKSIZE*256);
	hipMalloc((void **)&DEVICE_PQ_2, DMA_BLOCKSIZE*2);

	#ifdef ASYNC
	hipHostMalloc((void **)&HOST_DP_1, DMA_BLOCKSIZE*256);
	hipHostMalloc((void **)&HOST_PQ_1, DMA_BLOCKSIZE*2);
	
	hipHostMalloc((void **)&HOST_DP_2, DMA_BLOCKSIZE*256);
	hipHostMalloc((void **)&HOST_PQ_2, DMA_BLOCKSIZE*2);
	#endif
	
	hipError_t error_t;
	error_t = hipGetLastError();
	if( 0 != strcmp(hipGetErrorString(error_t), "no error") ){
		printf("Device allocation failed!\n");
		exit(1);
		}
	
	#ifdef DEBUG_LEVEL_8 
	printf("getting device memory : %s\n", hipGetErrorString(error_t) );
	#endif
	
	mem_tag = 1;
	}
}



/**
 * Free the memory from the device
 *
 * @returns			void
 */

extern void release_card_memory(void)
{
hipFree(DEVICE_DP_1);
hipFree(DEVICE_PQ_1);
	
hipFree(DEVICE_DP_2);
hipFree(DEVICE_PQ_2);
	
#ifdef ASYNC
hipFree(HOST_DP_1);
hipFree(HOST_PQ_1);
	
hipFree(HOST_DP_2);
hipFree(HOST_PQ_2);
#endif
}

