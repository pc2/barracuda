/**
 * \file
 * \brief	CUDA-Memorytest
 *
 * @author	Dominic Eschweiler weiler@upb.de
 *
 * Status	: STABLE \n
 * Date of creation : 20.11.2008
 */

/*****************************************************************
 *
 * Barracuda is a experimental microdriver extension to the 
 * linux-kernel that is able to outsource common functions to
 * the userspace. It was intensionally designed to accelerate
 * CPU-intensive Tasks on a GPU.
 *
 * Copyright (C) 2009 Dominic Eschweiler
 *
 * This program is free software; you can redistribute it and/or 
 * modify it under the terms of the GNU General Public License as 
 * published by the Free Software Foundation; either only GPLv2 - 
 * version 2 of the License.
 *
 * This program is distributed in the hope that it will be useful, 
 * but WITHOUT ANY WARRANTY; without even the implied warranty of 
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the 
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public 
 * License along with this program; 
 * if not, see <http://www.gnu.org/licenses/>.
 *
 *****************************************************************/

#include <stdlib.h>
#include <stdio.h>
#include <unistd.h>
#include <string.h>
#include <dirent.h>
#include <sys/stat.h>
#include <sys/file.h>
#include <sys/types.h>
#include <time.h>
#include <sys/time.h>
#include <math.h>
#include <sched.h>
#include <signal.h>
#include <sys/wait.h>
#include <syslog.h>

#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

int gen_pattern(const char *output_tag, unsigned char pattern, char *DEVICE_PQ, int freemem);

#define DMA_BLOCKSIZE 66560 * sizeof(unsigned char)
#define MAX_DISKS 255

/* This enables the debug statements */
//#define CUDACOPY


/**
 * Main Function of control
 *
 * @param argc		: # of arguments
 * @param **argv	: Array of arguments
 *
 * @returns			EXIT_FAILURE on error, EXIT_SUCCESS on no error
 */

int main( int argc, char *argv[] )
{
unsigned int freemem;
unsigned int total;
int deviceCount;
int device;
char name[256];
int ret;
int i;
int durations = 1;

char *DEVICE_PQ;
hipDevice_t 	dev;
struct hipDeviceProp_t prop;
hipError_t result;
	
hipInit(0);
printf("Barracuda CUDA memtest\n");

if(argc > 1){
	durations = atoi(argv[1]);
	if( (durations < 1) || (durations > 20) ){
		printf("Number of durations is to small or higher then 20!!!\n");
		return 1;
		}
	}

hipGetDeviceCount(&deviceCount);
for (device = 0; device < deviceCount; ++device) {
	hipSetDevice(device);
	
	hipDeviceGet(&dev, device);
	hipDeviceGetName((char *)&name, 256, dev);
	hipGetDeviceProperties(&prop, dev);
	
	printf("Getting device memory informations (device %d = %s).\n", device, name );
	
	hipDeviceTotalMem(&total, dev);
	freemem = total;
	
	printf("Total     : %d MB\n", ((total/1024)/1024) );
	
	result = hipMalloc((void **)&DEVICE_PQ, freemem);
	while( result == hipErrorOutOfMemory ){
		freemem = freemem - (1024*1024);
		result = hipMalloc((void**)&DEVICE_PQ, freemem);
		if(freemem < (1024*1024) ){
			printf("out of memmory!!!\n");
			return 1;
			}
		}

	printf("Free      : %d MB\n", ((freemem/1024)/1024) );
	printf("Durations : %d\n", durations);
	
	#ifdef CUDACOPY
	error_t = hipGetLastError();
	printf("%s\n", hipGetErrorString(error_t) );	
	#endif
	
	for(i=0; i<durations; i++){
		ret = gen_pattern("Generating pattern one                  : []", 85, DEVICE_PQ, freemem);
		if(ret == 1){
			return 1;
			}
	
		ret = gen_pattern("Generating pattern two (anti pattern)   : []", 170, DEVICE_PQ, freemem);
		if(ret == 1){
			return 1;
			}

		ret = gen_pattern("Generating pattern three                : []", 255, DEVICE_PQ, freemem);
		if(ret == 1){
			return 1;
			}

		ret = gen_pattern("Generating pattern four (anti pattern)  : []", 0, DEVICE_PQ, freemem);
		if(ret == 1){
			return 1;
			}
		
		}

	hipFree(DEVICE_PQ);
	
	printf("Test succeeded! -> This is a valid CUDA-device\n");
	}

return 0;
}



/**
 * Generate a pattern, copy to the GPU. After tthat copy the pattern back.
 *
 * @param *output_tag		: Local memory
 * @param pattern			: Bit-Pattern
 * @param *DEVICE_PQ		: Device Pointer
 * @param freemem			: Free card memory
 *
 * @returns			0 on Success, 1 on failure
 */

int gen_pattern(const char *output_tag, unsigned char pattern, char *DEVICE_PQ, int freemem)
{
int i,j;
unsigned char *pattern_pointer;
#ifdef CUDACOPY
	hipError_t error_t;
	const char *error;
#endif

printf("%s", output_tag);
pattern_pointer = (unsigned char *)malloc(sizeof(unsigned char) * freemem);

if(pattern == 0){
	pattern = 255;
	}

for(j=0; j<30; j++){
	printf("\b#]");
	fflush(stdout);

	memset( pattern_pointer, pattern, freemem);
	hipMemcpy(DEVICE_PQ, pattern_pointer, freemem, hipMemcpyHostToDevice);
	hipCtxSynchronize();

#ifdef CUDACOPY
	error_t = hipGetLastError();
	printf("\n%s\n", hipGetErrorString(error_t) );
#endif
		
	memset( pattern_pointer, 0, freemem);
	hipMemcpy(pattern_pointer, DEVICE_PQ, freemem, hipMemcpyDeviceToHost);

#ifdef CUDACOPY
	error_t = hipGetLastError();
	printf("\n%s\n", hipGetErrorString(error_t) );
#endif
	
	for(i = 0; i<freemem; i++){
		if(pattern_pointer[i] != pattern){
			printf("\nbad adress %u\n", i);
			hipFree(DEVICE_PQ);
			return 1;
			}
		}
	}

printf("\n");
free(pattern_pointer);
return 0;
}
