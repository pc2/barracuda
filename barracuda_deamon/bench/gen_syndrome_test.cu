#include "hip/hip_runtime.h"
/**
 * \file
 * \brief	This functions test the RS implmentations
 *
 * @author	Dominic Eschweiler weiler@upb.de
 *
 * Status	: STABLE\n
 * Date of creation : 14.8.2008
 *
 */

/*****************************************************************
 *
 * Barracuda is a experimental microdriver extension to the 
 * linux-kernel that is able to outsource common functions to
 * the userspace. It was intensionally designed to accelerate
 * CPU-intensive Tasks on a GPU.
 *
 * Copyright (C) 2009 Dominic Eschweiler
 *
 * This program is free software; you can redistribute it and/or 
 * modify it under the terms of the GNU General Public License as 
 * published by the Free Software Foundation; either only GPLv2 - 
 * version 2 of the License.
 *
 * This program is distributed in the hope that it will be useful, 
 * but WITHOUT ANY WARRANTY; without even the implied warranty of 
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the 
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public 
 * License along with this program; 
 * if not, see <http://www.gnu.org/licenses/>.
 *
 *****************************************************************/

#include <stdlib.h>
#include <stdio.h>
#include <unistd.h>
#include <string.h>
#include <dirent.h>
#include <sys/stat.h>
#include <sys/file.h>
#include <sys/types.h>
#include <time.h>
#include <sys/time.h>
#include <math.h>

# include "gen_syndrome_test.h"

void (*local_gen_syndrome)GEN_SYNDROME;	

/**
 * Benchmarks the pure Speed of all registered implementations
 *
 * @param number_of_disks	: # of data disks
 * @param **dptrs			: datapointer
 *
 * @returns		void
 */

HOST void compare_all_implementations_one_run(int number_of_disks, void **dptrs);

HOST unsigned long mill(void);

/**
 * Benchmarks the pure Speed of all registered implementations in a loops
 *
 * @param GEN_SYNDROME				: function pointer
 * @param **implemenatation_names	: related names of each function
 *
 * @param number_of_implementations	: # of implementations
 *
 * @returns		void
 */

HOST void compare_all_implementations(	void (*gen_syndrome_list[])GEN_SYNDROME,
								 			char **implemenatation_names,
								 			int number_of_implementations)
{
printf("\"X\" ; \"Y\"\n");	
int i = 0;
int j = 0;

local_gen_syndrome = gen_syndrome_list[number_of_implementations];
void **dptrs;
	
/** 
 * Test begins with 5 Disks, because RAID6 is not worth for using with a lesser 
 * number of devices
 */
dptrs = allocate_host_example_dpointer( PAGE_SIZE, 66 );

for(j=5; j<=64; j++){
	/* Do every test 10 times for upt 64 disks*/
	for(i=0; i<10; i++){
		compare_all_implementations_one_run(j, dptrs);
		}
	}
	
deallocate_host_example_dpointer( 66, dptrs );
}



/**
 * Benchmarks the pure Speed of all registered implementations
 *
 * @param number_of_disks	: # of data disks
 * @param **dptrs			: datapointer
 *
 * @returns		void
 */

HOST void compare_all_implementations_one_run(int number_of_disks, void **dptrs)
{

/** 
 * clocks() based version, please handle with care, because it handles only
 * cpu-time 
 */
/*
unsigned long j;
unsigned long clocks_timer;
unsigned long clocks_tmp_timer;

j = 0;
clocks_timer = 0;
	
while(clocks_timer < 2*CLOCKS_PER_SEC){
	clocks_tmp_timer = clock();
		local_gen_syndrome(number_of_disks, PAGE_SIZE, dptrs);
	clocks_timer = clocks_timer = clocks_timer + (clock() - clocks_tmp_timer);
	j++;
	}

printf("%d ; %u\n", number_of_disks, (unsigned long)((PAGE_SIZE*j)/(clocks_timer/CLOCKS_PER_SEC) ) );
*/

/* high gtd time version with seconds*/
/*
double t1, t2;
int i;
unsigned long ticks;
	
//int dur = 4096 / number_of_disks;
int dur = 1000;
	
t1 = gtd_second();
for(i=0; i<dur; i++){
	local_gen_syndrome(number_of_disks, PAGE_SIZE, dptrs);
	}
t2 = gtd_second();
t2 = t2 - t1;

ticks = (unsigned long)((PAGE_SIZE*dur)/t2);
printf("%d ; %u\n", number_of_disks, ticks);
*/
	
/* high resolution gtd time version */
unsigned t1, t2;
int i;
unsigned long ticks;
	
//int dur = 4096 / number_of_disks;
int dur = 1000;
	
t1 = mill();
for(i=0; i<dur; i++){
	local_gen_syndrome(number_of_disks, PAGE_SIZE, dptrs);
	}
t2 = mill();
t2 = t2 - t1;

ticks = (unsigned long)((PAGE_SIZE*dur)/t2);
ticks = ticks * 1000;
printf("%d ; %u\n", number_of_disks, ticks);

}


HOST unsigned long mill(void)
{
struct timezone tz;
struct timeval t;
gettimeofday(&t, &tz);

return (long)( (t.tv_sec*1000) + (t.tv_usec/1000) );
}

