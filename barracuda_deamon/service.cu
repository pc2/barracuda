#include "hip/hip_runtime.h"
/**
 * \file
 * \brief	library for helper functions
 *
 * @author	Dominic Eschweiler weiler@upb.de
 *
 * Status	: STABLE \n
 * Date of creation : 19.5.2008
 *
 */

/*****************************************************************
 *
 * Barracuda is a experimental microdriver extension to the 
 * linux-kernel that is able to outsource common functions to
 * the userspace. It was intensionally designed to accelerate
 * CPU-intensive Tasks on a GPU.
 *
 * Copyright (C) 2009 Dominic Eschweiler
 *
 * This program is free software; you can redistribute it and/or 
 * modify it under the terms of the GNU General Public License as 
 * published by the Free Software Foundation; either only GPLv2 - 
 * version 2 of the License.
 *
 * This program is distributed in the hope that it will be useful, 
 * but WITHOUT ANY WARRANTY; without even the implied warranty of 
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the 
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public 
 * License along with this program; 
 * if not, see <http://www.gnu.org/licenses/>.
 *
 *****************************************************************/

#include <stdlib.h>
#include <stdio.h>
#include <unistd.h>
#include <string.h>
#include <dirent.h>
#include <math.h>
#include <time.h>

#include <sys/time.h>
#include <sys/mman.h>
#include <sys/stat.h>
#include <sys/file.h>
#include <sys/types.h>

#include <linux/types.h>

#include "service.h"

static int NUMBER_OF_CPUS_INSTALLED = 1;

/**
 * The gtd_second function returns the amount of time, where the process 
 * is running. It uses the propper glibc function gettimeofday() which
 * which extracts from the RTC
 *
 * @returns		Time
 */

HOST double gtd_second(void)
{
	struct timezone tz;
	struct timeval t;
	gettimeofday(&t, &tz);

	return (double) t.tv_sec + ((double)t.tv_usec/1e6);
}



/**
 * The second function returns the amount of time, where the process 
 * is running. It uses the propper glibc function gettimeofday() which
 * extracts from the RTC
 *
 * @returns		Time
 */

HOST double second(void)
{
double secs;
clock_t Time;
Time = clock();
secs = (double)Time / (double)CLOCKS_PER_SEC;
return secs ;
}



/**
 * Generates a test datapointer for the gen_syndrome function, which can be used
 * by the validator or benchmarking functions.
 *
 * @param bytes				: # of bytes
 * @param number_of_disks	: Number of virtual data disks
 *
 * @returns				Datapointers were the gft an syndrome gets saved in
 */

HOST void **allocate_host_example_dpointer( int bytes, int number_of_disks )
{
/* Variables */
int i;
void **dptrs;
	
#ifdef DEBUG_LEVEL_2
	printf("-= DEBUG 2 =- \n");
	print_dpointer( number_of_disks, PAGE_SIZE, dptrs);
	printf("-= DEBUG 2 =- \n");
#endif
	
/* Set virtual disk data */
dptrs = (void **)malloc( (number_of_disks)*sizeof(void *) );
for ( i = 0; i < number_of_disks; i++ ){
	dptrs[i] = (void *)malloc(bytes*sizeof(u8));
	}

return dptrs;
}



/**
 * Deallocates the example dpointer
 *
 * @param number_of_disks	: Number of virtual data disks
 * @param **dptrs			: datapointer
 *
 * @returns				Datapointers were the gft an syndrome gets saved in
 */

HOST void deallocate_host_example_dpointer( int number_of_disks, void **dptrs )
{
/* Variables */
int i;

/* Free virtual disk data */
for ( i = 0 ; i < number_of_disks ; i++ ){
	free(dptrs[i]);
	}
free(dptrs);
}



/**
 * Prints a dpointer to the console.
 *
 * @param disks			: # of disks
 * @param bytes			: # of bytes
 * @param **ptrs		: Datapointers were the gft an syndrome gets saved in
 *
 * @returns		# of disks
 */

HOST void print_dpointer(int disks, int bytes, void **ptrs)
{
int i;
	
u8 **dptrs = (u8 **)ptrs;
u8 *xor_d  = dptrs[disks];
u8 *syn_d  = dptrs[disks+1];

for(i=0; i < bytes; i++){
	printf("%d %d ", xor_d[i], syn_d[i]);
	}

printf("\n");

}



/**
 * Inititalize the generation of additional system variables. This variables can
 * be get with the following functions :
 * get_number_of_phys_cpus() : Get the number of SMP Processors in your system
 *
 * @returns	 void
 */

void set_internal_vars()
{
FILE *fpointer;
char buffer[256];
char *n_buffer;

/* initially set the number of cpus to 1 */
NUMBER_OF_CPUS_INSTALLED = 1;
	
/* open the sysfs file that shows the number of cpus */
fpointer = fopen("/sys/devices/system/cpu/online", "r");	
if(fpointer == NULL){
	/**
	 * The CPU info dir exists only if there are multiple CPUs. Therfore, if the
	 * opening fails, the default value 1 is used. 
	 */
	return;
	}
else{
	/**
	 * In this case, the infodir exists and the value of cpus can be simply
	 * extracted from 
	 */
	fgets( (char *)&buffer, 256, fpointer);
	strtok((char *)&buffer, "-");
	n_buffer = strtok(NULL, "-");
	
	NUMBER_OF_CPUS_INSTALLED = atoi(n_buffer)+1;
	
	fclose(fpointer);
	}
	
}



/**
 * This function returns the number of physical CPUs in the system. Mainly it 
 * returns the global variable NUMBER_OF_CPUS_INSTALLED, which is 1 per default
 * and set to the right number of CPUs by the function set_internal_vars().
 *
 * @returns	 int : number of CPUs in the system
 */

int get_number_of_phys_cpus()
{
return NUMBER_OF_CPUS_INSTALLED;
}
